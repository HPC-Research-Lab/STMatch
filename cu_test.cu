#include <string>
#include <iostream>
#include "src/gpu_match.cuh"

using namespace std;
using namespace libra;

int main(int argc, char* argv[]) {

  libra::GraphPreprocessor g(argv[1]);
  libra::PatternPreprocessor p(argv[2]);

  // copy graph and pattern to GPU global memory
  Graph* gpu_graph = g.to_gpu();
  Pattern* gpu_pattern = p.to_gpu();
  JobQueue* gpu_queue = JobQueuePreprocessor(g.g, p.pat).to_gpu();
  CallStack* gpu_callstack;

  // allocate the callstack for all warps in global memory
  graph_node_t* slot_storage;
  hipMalloc(&slot_storage, sizeof(graph_node_t) * NWARPS_TOTAL * PAT_SIZE * PAT_SIZE * GRAPH_DEGREE);
  cout << "global memory usage: " << sizeof(graph_node_t) * NWARPS_TOTAL * PAT_SIZE * PAT_SIZE * GRAPH_DEGREE / 1024.0 / 1024 / 1024 << " GB" << endl;

  std::vector<CallStack> stk(NWARPS_TOTAL);

  for (int i = 0; i < NWARPS_TOTAL; i++) {
    auto& s = stk[i];
    memset(s.iter, 0, sizeof(s.iter));
    memset(s.slot_size, 0, sizeof(s.slot_size));
    s.slot_storage = (graph_node_t(*)[PAT_SIZE][GRAPH_DEGREE])((char*)slot_storage + i * sizeof(graph_node_t) * PAT_SIZE * PAT_SIZE * GRAPH_DEGREE);
  }
  hipMalloc(&gpu_callstack, NWARPS_TOTAL * sizeof(CallStack));
  hipMemcpy(gpu_callstack, stk.data(), sizeof(CallStack) * NWARPS_TOTAL, hipMemcpyHostToDevice);

  size_t* gpu_res;
  hipMalloc(&gpu_res, sizeof(size_t) * NWARPS_TOTAL);
  hipMemset(gpu_res, 0, sizeof(size_t) * NWARPS_TOTAL);
  size_t* res = new size_t[NWARPS_TOTAL];


  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  cout << "shared memory usage: " << sizeof(Graph) << " " << sizeof(Pattern) << " " << sizeof(JobQueue) << " " << sizeof(CallStack) * NWARPS_PER_BLOCK << " " << NWARPS_PER_BLOCK * 33 * sizeof(int) << " Bytes" << endl;

  _parallel_match << <GRID_DIM, BLOCK_DIM >> > (gpu_graph, gpu_pattern, gpu_callstack, gpu_queue, gpu_res);


  hipEventRecord(stop);

  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("matching time: %f ms\n", milliseconds);

  hipMemcpy(res, gpu_res, sizeof(size_t) * NWARPS_TOTAL, hipMemcpyDeviceToHost);

  size_t tot_count = 0;
  for (int i=0; i<NWARPS_TOTAL; i++) tot_count += res[i];

  cout << "count: " << tot_count << endl;
  return 0;
}
