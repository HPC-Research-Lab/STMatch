#include <string>
#include <iostream>
#include "src/gpu_match.cuh"

using namespace std;
using namespace libra;

int main(int argc, char* argv[]) {

  hipSetDevice(0);

  libra::GraphPreprocessor g(argv[1]);
  libra::PatternPreprocessor p(argv[2]);

  // copy graph and pattern to GPU global memory
  Graph* gpu_graph = g.to_gpu();
  Pattern* gpu_pattern = p.to_gpu();
  JobQueue* gpu_queue = JobQueuePreprocessor(g.g, p).to_gpu();
  CallStack* gpu_callstack;

  // allocate the callstack for all warps in global memory
  graph_node_t* slot_storage;
  hipMalloc(&slot_storage, sizeof(graph_node_t) * NWARPS_TOTAL * MAX_SLOT_NUM * UNROLL * GRAPH_DEGREE);
  //cout << "global memory usage: " << sizeof(graph_node_t) * NWARPS_TOTAL * MAX_SLOT_NUM * UNROLL * GRAPH_DEGREE / 1024.0 / 1024 / 1024 << " GB" << endl;

  std::vector<CallStack> stk(NWARPS_TOTAL);

  for (int i = 0; i < NWARPS_TOTAL; i++) {
    auto& s = stk[i];
    memset(s.iter, 0, sizeof(s.iter));
    memset(s.slot_size, 0, sizeof(s.slot_size));
    s.slot_storage = (graph_node_t(*)[UNROLL][GRAPH_DEGREE])((char*)slot_storage + i * sizeof(graph_node_t) * MAX_SLOT_NUM * UNROLL * GRAPH_DEGREE);
  }
  hipMalloc(&gpu_callstack, NWARPS_TOTAL * sizeof(CallStack));
  hipMemcpy(gpu_callstack, stk.data(), sizeof(CallStack) * NWARPS_TOTAL, hipMemcpyHostToDevice);

  size_t* gpu_res;
  hipMalloc(&gpu_res, sizeof(size_t) * NWARPS_TOTAL);
  hipMemset(gpu_res, 0, sizeof(size_t) * NWARPS_TOTAL);
  size_t* res = new size_t[NWARPS_TOTAL];

  int* idle_warps;
  hipMalloc(&idle_warps, sizeof(int) * GRID_DIM);
  hipMemset(idle_warps, 0, sizeof(int) * GRID_DIM);

  int* idle_warps_count;
  hipMalloc(&idle_warps_count, sizeof(int));
  hipMemset(idle_warps_count, 0, sizeof(int));

  int* global_mutex;
  hipMalloc(&global_mutex, sizeof(int) * GRID_DIM);
  hipMemset(global_mutex, 0, sizeof(int) * GRID_DIM);

  bool* stk_valid;
  hipMalloc(&stk_valid, sizeof(bool) * GRID_DIM);
  hipMemset(stk_valid, 0, sizeof(bool) * GRID_DIM);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  //cout << "shared memory usage: " << sizeof(Graph) << " " << sizeof(Pattern) << " " << sizeof(JobQueue) << " " << sizeof(CallStack) * NWARPS_PER_BLOCK << " " << NWARPS_PER_BLOCK * 33 * sizeof(int) << " Bytes" << endl;

  _parallel_match << <GRID_DIM, BLOCK_DIM >> > (gpu_graph, gpu_pattern, gpu_callstack, gpu_queue, gpu_res, idle_warps, idle_warps_count, global_mutex);


  hipEventRecord(stop);

  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  //printf("matching time: %f ms\n", milliseconds);

  hipMemcpy(res, gpu_res, sizeof(size_t) * NWARPS_TOTAL, hipMemcpyDeviceToHost);

  size_t tot_count = 0;
  for (int i=0; i<NWARPS_TOTAL; i++) tot_count += res[i];
  printf("%s\t%f\t%lu\n", argv[2], milliseconds, tot_count);
  //cout << "count: " << tot_count << endl;
  return 0;
}
