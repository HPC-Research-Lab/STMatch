#include <string>
#include <iostream>
#include "src/gpu_match.cuh"

using namespace std;
using namespace libra;

int main(int argc, char* argv[]) {

  libra::GraphPreprocessor g(argv[1]);
  libra::PatternPreprocessor p(argv[2]);

  // copy graph and pattern to GPU global memory
  Graph* gpu_graph = g.to_gpu();
  Pattern* gpu_pattern = p.to_gpu();
  JobQueue* gpu_queue = JobQueuePreprocessor(g.g).to_gpu();
  CallStack* gpu_callstack;

  // allocate the callstack for all warps in global memory
  graph_node_t* slot_storage;
  hipMalloc(&slot_storage, sizeof(graph_node_t) * NWARPS_TOTAL * PAT_SIZE * PAT_SIZE * GRAPH_DEGREE);

  std::vector<CallStack> stk(NWARPS_TOTAL);

  for (int i = 0; i < NWARPS_TOTAL; i++) {
    auto& s = stk[i];
    memset(s.path, 0, sizeof(s.path));
    memset(s.iter, 0, sizeof(s.iter));
    memset(s.slot_size, 0, sizeof(s.slot_size));
    s.slot_storage = (graph_node_t(*)[PAT_SIZE][GRAPH_DEGREE])(slot_storage + i * sizeof(graph_node_t) * PAT_SIZE * PAT_SIZE * GRAPH_DEGREE);
  }
  hipMalloc(&gpu_callstack, NWARPS_TOTAL * sizeof(CallStack));
  hipMemcpy(gpu_callstack, stk.data(), sizeof(CallStack) * NWARPS_TOTAL, hipMemcpyHostToDevice);


  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  std::cout << "shared memory usage: " << sizeof(Graph) << " " << sizeof(Pattern) << " " << sizeof(JobQueue) << " " << sizeof(CallStack) * NWARPS_PER_BLOCK << std::endl;

  _parallel_match << <GRID_DIM, BLOCK_DIM >> > (gpu_graph, gpu_pattern, gpu_callstack, gpu_queue);

  hipDeviceSynchronize();

  hipEventRecord(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("matching time: %f ms\n", milliseconds);;
  return 0;
}
