#include <string>
#include <iostream>
#include <unistd.h>
#include <omp.h>
#include "src/gpu_match.cuh"

using namespace std;
using namespace STMatch;

void printfLocalProfile(ProfInfo& profile){

  double gridTotal = 0;
  for(int i=0; i<GRID_DIM; i++){
    double blockTotal = 0; 
    for(int j=0; j<BLOCK_DIM; j+=WARP_SIZE){
        blockTotal+=profile.clk[i][j];
    }
    gridTotal+=(blockTotal/NWARPS_PER_BLOCK);
  }
  //printf("%f\n", gridTotal/GRID_DIM);

  double memTotal = 0; 
  for(int i=0; i<GRID_DIM; i++){
    for(int j=0; j<NWARPS_PER_BLOCK; j++){
        memTotal+=profile.localMemStorage[i][j];
    }
  }
  printf("%f\n", memTotal/GRID_DIM);

}

void printfUtilization(ProfInfo& profile){
  uint64_t totalThreadUsed = 0;
  uint64_t busyThreadUsed = 0;
  for(int i=0; i<GRID_DIM; i++){
    for(int j=0; j<NWARPS_PER_BLOCK; j++){
      totalThreadUsed+=profile.totalThreadUsed[i][j];
      busyThreadUsed+=profile.busyThreadUsed[i][j];
    }
  }
   printf("%f\n", (double)busyThreadUsed/(double)totalThreadUsed);
}


void printfGlobalProfile(ProfInfo& profile){
  //------------------Global-----------
  double gridTotal = 0;
  for(int i=0; i<GRID_DIM; i++){
    long long int blockMax = 0; 
    for(int j=0; j<BLOCK_DIM; j+=WARP_SIZE){
       if(profile.clk[i][j]>blockMax) blockMax = profile.clk[i][j];
    }
    gridTotal+=blockMax;
  }
  //printf("%f\n", gridTotal/GRID_DIM);


  size_t storgeTotal = 0;
  size_t stkTotal = 0;
  for(int i=0; i<GRID_DIM; i++){
    for(int j=0; j<NWARPS_PER_BLOCK; j++){
      storgeTotal+=profile.globalMemStorage[i][j];
      stkTotal+=profile.globalMemStk[i][j];
    }
  }
  printf("%lu\t%lu\n", storgeTotal, stkTotal);

}


int main(int argc, char* argv[]) {
  STMatch::GraphPreprocessor g(argv[1]);
  STMatch::PatternPreprocessor p(argv[2]);


  //JobQueue* gpu_queue[NUM_GPU] = JobQueuePreprocessor(g.g, p).distributedQueue();
  Graph* gpu_graph[NUM_GPU];
  Pattern* gpu_pattern[NUM_GPU];
  JobQueue* gpu_queue[NUM_GPU];
  CallStack* gpu_callstack[NUM_GPU];
  graph_node_t* slot_storage[NUM_GPU];
  size_t* gpu_res[NUM_GPU];

  int* idle_warps[NUM_GPU];
  int* idle_warps_count[NUM_GPU];
  int* global_mutex[NUM_GPU];
  bool* stk_valid[NUM_GPU];

  for(int gpuIdx=0; gpuIdx<NUM_GPU; gpuIdx++){
    hipSetDevice(gpuIdx);
    
    gpu_graph[gpuIdx] = g.to_gpu();
    gpu_pattern[gpuIdx] = p.to_gpu();
    gpu_queue[gpuIdx] = JobQueuePreprocessor(g.g, p).to_gpu(gpuIdx);
    hipMalloc(&slot_storage[gpuIdx], sizeof(graph_node_t) *  NWARPS_TOTAL * MAX_SLOT_NUM * UNROLL * GRAPH_DEGREE);
    std::vector<CallStack> stk(NWARPS_TOTAL);
    for (int i = 0; i < NWARPS_TOTAL; i++) {
      auto& s = stk[i];
      memset(s.iter, 0, sizeof(s.iter));
      memset(s.slot_size, 0, sizeof(s.slot_size));
      s.slot_storage = (graph_node_t(*)[UNROLL][GRAPH_DEGREE])((char*)slot_storage[gpuIdx] + i * sizeof(graph_node_t) * MAX_SLOT_NUM * UNROLL * GRAPH_DEGREE);
    }
    hipMalloc(&gpu_callstack[gpuIdx], NWARPS_TOTAL * sizeof(CallStack));
    hipMemcpy(gpu_callstack[gpuIdx], stk.data(), sizeof(CallStack) * NWARPS_TOTAL, hipMemcpyHostToDevice);

    hipMalloc(&gpu_res[gpuIdx], sizeof(size_t) * NWARPS_TOTAL);
    hipMemset(gpu_res[gpuIdx], 0, sizeof(size_t) * NWARPS_TOTAL);

    hipMalloc(&idle_warps[gpuIdx], sizeof(int) * GRID_DIM);
    hipMemset(idle_warps[gpuIdx], 0, sizeof(int) * GRID_DIM);

    hipMalloc(&idle_warps_count[gpuIdx], sizeof(int));
    hipMemset(idle_warps_count[gpuIdx], 0, sizeof(int));

    hipMalloc(&global_mutex[gpuIdx], sizeof(int) * GRID_DIM);
    hipMemset(global_mutex[gpuIdx], 0, sizeof(int) * GRID_DIM);

    hipMalloc(&stk_valid[gpuIdx], sizeof(bool) * GRID_DIM);
    hipMemset(stk_valid[gpuIdx], 0, sizeof(bool) * GRID_DIM);

  }
  
  size_t* res = new size_t[NWARPS_TOTAL];
  hipEvent_t start[NUM_GPU], stop[NUM_GPU];
  float milliseconds[NUM_GPU];
  
//--------------------  
  #pragma omp parallel for num_threads(NUM_GPU)
  for(int i=0; i<NUM_GPU; i++) {
      hipSetDevice(i);
      hipEventCreate(&start[i]);
      hipEventCreate(&stop[i]);
      hipEventRecord(start[i]);
      _parallel_match << <GRID_DIM, BLOCK_DIM>> > (gpu_graph[i], gpu_pattern[i], gpu_callstack[i], gpu_queue[i], gpu_res[i], idle_warps[i], idle_warps_count[i], global_mutex[i], i);
      hipEventRecord(stop[i]);
      hipEventSynchronize(stop[i]);
      hipEventElapsedTime(&milliseconds[i], start[i], stop[i]);
   }
   //printf("All Finished\n");



  float maxGPU = 0;
  uint64_t finalCount =0;
  for(int i=0; i<NUM_GPU; i++) {
    hipMemcpy(res, gpu_res[i], sizeof(size_t) * NWARPS_TOTAL, hipMemcpyDeviceToHost);
    size_t tot_count = 0;
    for (int j=0; j<NWARPS_TOTAL; j++) {
      tot_count += res[j];
    }
    if(milliseconds[i]>maxGPU) maxGPU = milliseconds[i];
    finalCount+=tot_count;

    //printf("%f\t", milliseconds[i]);
  }
  printf("%f\t%lu\n", maxGPU, finalCount);

  return 0;
}
