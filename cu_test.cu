#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include "src/gpu_match.cuh"

using namespace std;
using namespace STMatch;

void printfProfile(ProfInfo& profile){
  for(int i=0; i<1; i++){
    long long int max = 0;
    long long int total = 0; 
    for(int j=0; j<BLOCK_DIM; j+=WARP_SIZE){
      total+= profile.clk[i][j];
      if(profile.clk[i][j]>max) max = profile.clk[i][j];
      //printf("%lld\n", profile.clk[i][j]);
    }
    printf("clk:\t%f\t%lld\t", (double)total/(double)NWARPS_PER_BLOCK, max);
  }

  for(int i=0; i<1; i++){
    size_t total = 0;
    for(int j=0; j<NWARPS_PER_BLOCK; j++){
      total+=profile.localMemStorage[i][j];
      //printf("%lld\n", profile.clk[i][j]);
    }
    printf("%lu\t", total);
  }



  uint64_t totalThreadUsed = 0;
  uint64_t busyThreadUsed = 0;
  for(int i=0; i<GRID_DIM; i++){
    for(int j=0; j<NWARPS_PER_BLOCK; j++){
      totalThreadUsed+=profile.totalThreadUsed[i][j];
      busyThreadUsed+=profile.busyThreadUsed[i][j];
    }
  }
   printf("%f\n", (double)busyThreadUsed/(double)totalThreadUsed);

  //------------------Global-----------

  double gridTotal = 0;
  double gridMax = 0;
  for(int i=0; i<GRID_DIM; i++){
    long long int blockTotal = 0; 
    for(int j=0; j<BLOCK_DIM; j+=WARP_SIZE){
        blockTotal+= profile.clk[i][j];
    }
    double blockAvg = (double)blockTotal/NWARPS_PER_BLOCK;

    gridTotal+=blockAvg;
    if(blockAvg>gridMax) gridMax = blockAvg;
  }
  double gridAvg = gridTotal/GRID_DIM;
  //printf("%f\t%f\n", gridAvg, gridMax);


}


int main(int argc, char* argv[]) {

  hipSetDevice(0);

  STMatch::GraphPreprocessor g(argv[1]);
  STMatch::PatternPreprocessor p(argv[2]);

  // copy graph and pattern to GPU global memory
  Graph* gpu_graph = g.to_gpu();
  Pattern* gpu_pattern = p.to_gpu();
  JobQueue* gpu_queue = JobQueuePreprocessor(g.g, p).to_gpu();
  CallStack* gpu_callstack;

  // allocate the callstack for all warps in global memory
  graph_node_t* slot_storage;
  hipMalloc(&slot_storage, sizeof(graph_node_t) * NWARPS_TOTAL * MAX_SLOT_NUM * UNROLL * GRAPH_DEGREE);
  //cout << "global memory usage: " << sizeof(graph_node_t) * NWARPS_TOTAL * MAX_SLOT_NUM * UNROLL * GRAPH_DEGREE / 1024.0 / 1024 / 1024 << " GB" << endl;

  std::vector<CallStack> stk(NWARPS_TOTAL);

  for (int i = 0; i < NWARPS_TOTAL; i++) {
    auto& s = stk[i];
    memset(s.iter, 0, sizeof(s.iter));
    memset(s.slot_size, 0, sizeof(s.slot_size));
    s.slot_storage = (graph_node_t(*)[UNROLL][GRAPH_DEGREE])((char*)slot_storage + i * sizeof(graph_node_t) * MAX_SLOT_NUM * UNROLL * GRAPH_DEGREE);
  }
  hipMalloc(&gpu_callstack, NWARPS_TOTAL * sizeof(CallStack));
  hipMemcpy(gpu_callstack, stk.data(), sizeof(CallStack) * NWARPS_TOTAL, hipMemcpyHostToDevice);

  size_t* gpu_res;
  hipMalloc(&gpu_res, sizeof(size_t) * NWARPS_TOTAL);
  hipMemset(gpu_res, 0, sizeof(size_t) * NWARPS_TOTAL);
  size_t* res = new size_t[NWARPS_TOTAL];

  int* idle_warps;
  hipMalloc(&idle_warps, sizeof(int) * GRID_DIM);
  hipMemset(idle_warps, 0, sizeof(int) * GRID_DIM);

  int* idle_warps_count;
  hipMalloc(&idle_warps_count, sizeof(int));
  hipMemset(idle_warps_count, 0, sizeof(int));

  int* global_mutex;
  hipMalloc(&global_mutex, sizeof(int) * GRID_DIM);
  hipMemset(global_mutex, 0, sizeof(int) * GRID_DIM);

  bool* stk_valid;
  hipMalloc(&stk_valid, sizeof(bool) * GRID_DIM);
  hipMemset(stk_valid, 0, sizeof(bool) * GRID_DIM);

  ProfInfo* prof_info;
  hipMalloc(&prof_info, sizeof(ProfInfo));
  hipMemset(prof_info, 0, sizeof(ProfInfo));


  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  //cout << "shared memory usage: " << sizeof(Graph) << " " << sizeof(Pattern) << " " << sizeof(JobQueue) << " " << sizeof(CallStack) * NWARPS_PER_BLOCK << " " << NWARPS_PER_BLOCK * 33 * sizeof(int) << " Bytes" << endl;

  _parallel_match << <GRID_DIM, BLOCK_DIM >> > (gpu_graph, gpu_pattern, gpu_callstack, gpu_queue, gpu_res, idle_warps, idle_warps_count, global_mutex, prof_info);


  hipEventRecord(stop);

  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  //printf("matching time: %f ms\n", milliseconds);

  hipMemcpy(res, gpu_res, sizeof(size_t) * NWARPS_TOTAL, hipMemcpyDeviceToHost);


  ProfInfo cpuProfInfo;
  hipMemcpy(&cpuProfInfo, prof_info, sizeof(ProfInfo), hipMemcpyDeviceToHost);

  printfProfile(cpuProfInfo);

  size_t tot_count = 0;
  for (int i=0; i<NWARPS_TOTAL; i++) tot_count += res[i];
  //printf("%s\t%f\t%lu\n", argv[2], milliseconds, tot_count);
  //cout << "count: " << tot_count << endl;
  return 0;
}
