/*
 * cuTS:  Scaling Subgraph Isomorphism on Distributed Multi-GPU Systems Using
 *        Trie Based Data Structure
 *
 * Copyright (C) 2021 APPL Laboratories (aravind_sr@outlook.com)
 *
 * This software is available under the MIT license, a copy of which can be
 * found in the file 'LICENSE' in the top-level directory.
 *
 * For further information contact:
 *   (1) Lizhi Xiang (lizhi.xiang@wsu.edu)
 *   (2) Aravind Sukumaran-Rajam (aravind_sr@outlook.com)
 *
 * The citation information is provided in the 'README' in the top-level
 * directory.
 */
#include "../inc/gpu_memory_allocation.h"
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cout<<hipGetErrorString(code)<<std::endl;
        exit(-1);
    }
}
void malloc_graph_gpu_memory(Graph &g,G_pointers &p){
    chkerr(hipMalloc(&(p.neighbors),g.neighbors_offset[g.V]*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.neighbors,g.neighbors,g.neighbors_offset[g.V]*sizeof(unsigned int),hipMemcpyHostToDevice));
    chkerr(hipMalloc(&(p.neighbors_offset),(g.V+1)*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.neighbors_offset,g.neighbors_offset,(g.V+1)*sizeof(unsigned int),hipMemcpyHostToDevice));
    chkerr(hipMalloc(&(p.signatures),(g.V)*sizeof(unsigned int)*Signature_Properties));
    chkerr(hipMemcpy(p.signatures,g.signatures,(g.V)*sizeof(unsigned int)*Signature_Properties,hipMemcpyHostToDevice));
    chkerr(hipMalloc(&(p.r_neighbors),g.r_neighbors_offset[g.V]*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.r_neighbors,g.r_neighbors,g.r_neighbors_offset[g.V]*sizeof(unsigned int),hipMemcpyHostToDevice));
    chkerr(hipMalloc(&(p.r_neighbors_offset),(g.V+1)*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.r_neighbors_offset,g.r_neighbors_offset,(g.V+1)*sizeof(unsigned int),hipMemcpyHostToDevice));
    p.V = g.V;
}
void malloc_query_constraints_gpu_memory(Graph &g,C_pointers &p){
    chkerr(hipMalloc(&(p.parents),g.parents_offset[g.V]*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.parents,g.parents,g.parents_offset[g.V]*sizeof(unsigned int),hipMemcpyHostToDevice));
    chkerr(hipMalloc(&(p.parents_offset),(g.V+1)*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.parents_offset,g.parents_offset,(g.V+1)*sizeof(unsigned int),hipMemcpyHostToDevice));
    chkerr(hipMalloc(&(p.children),g.children_offset[g.V]*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.children,g.children,g.children_offset[g.V]*sizeof(unsigned int),hipMemcpyHostToDevice));
    chkerr(hipMalloc(&(p.children_offset),(g.V+1)*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.children_offset,g.children_offset,(g.V+1)*sizeof(unsigned int),hipMemcpyHostToDevice));
    chkerr(hipMalloc(&(p.order_sqeuence),(g.V)*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.order_sqeuence,g.order_sequence,(g.V)*sizeof(unsigned int),hipMemcpyHostToDevice));
}
void malloc_other_searching_gpu_memory(S_pointers &p,unsigned int workers,unsigned int max_nodes){
    chkerr(hipMallocManaged(&(p.lengths),(max_nodes+1)*sizeof(unsigned int)));
    chkerr(hipMemset(p.lengths,0,(max_nodes+1)*sizeof(unsigned int)));
    chkerr(hipMalloc(&(p.helper_buffer1),workers*HelperSize*sizeof(unsigned int)));
    chkerr(hipMalloc(&(p.helper_buffer2),workers*HelperSize*sizeof(unsigned int)));
    unsigned int remaining_words = GPU_TABLE_LIMIT - workers*HelperSize;
    unsigned long long int table_size = remaining_words * sizeof(unsigned int);
    chkerr(hipMalloc(&(p.results_table),table_size));
    
    //printf("GPU_TABLE_LIMIT:%ld, workers:%ld, HelperSize:%ld,  table_size:%ld\n", GPU_TABLE_LIMIT, workers, HelperSize, table_size);
    chkerr(hipMalloc(&(p.indexes_table),table_size));
    unsigned long long int cpu_table_size = CPU_FINAL_TABLE_SIZE * sizeof(unsigned int);
    chkerr(hipMallocManaged(&(p.final_results_table),cpu_table_size));
    chkerr(hipMallocManaged(&(p.final_count),sizeof(unsigned long long int)));
    p.final_count[0] = 0;
    chkerr(hipMalloc(&(p.write_pos),sizeof(unsigned long long int)));
    chkerr(hipMemset(p.write_pos,0,sizeof(unsigned long long int)));
    unsigned long long int row_ptrs_size = FINAL_RESULTS_ROW_PTR_SIZE * sizeof(unsigned long long int);
    chkerr(hipMallocManaged(&(p.final_results_row_ptrs),row_ptrs_size));
    chkerr(hipMallocManaged(&(p.indexes_pos),sizeof(unsigned long long int)));
    chkerr(hipMemset(p.indexes_pos,0,sizeof(unsigned long long int)));
}
