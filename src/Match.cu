#include "hip/hip_runtime.h"
#include "Match.cuh"
#include <hip/hip_runtime.h>

namespace libra {

  // TODO: implement intersection and difference
  template<typename DATA_T, typename SIZE_T>
  __device__ void intersection(DATA_T* set1, DATA_T* set2, SIZE_T set1_size, SIZE_T set2_size, DATA_T ub) {

  }

  template<typename DATA_T, typename SIZE_T>
  __device__ void difference(DATA_T* set1, DATA_T* set2, SIZE_T set1_size, SIZE_T set2_size, DATA_T ub) {

  }

  __device__ void lock(int* mutex) {
    while (atomicCAS(mutex, 0, 1) != 0);
  }

  __device__ void unlock(int* mutex) {
    atomicExch(mutex, 0);
  }

  __device__ void get_job(JobQueue* q, graph_node_t& cur_job, graph_node_t& njobs) {
    lock(&(q->mutex));
    graph_node_t cur_pos = q->cur;
    q->cur += JOB_CHUNK_SIZE;
    if (q->cur > q->length) q->cur = q->length;
    njobs = q->cur - cur_pos;
    unlock(&(q->mutex));
  }

  __device__ void match(Graph* g, Pattern* pat, CallStack* stk, JobQueue* q, size_t* count) {
    graph_node_t level = 0;

    while (true) {

      if (level < pat->nnodes - 1) {
        if (stk->slot_size[level][0] == 0) {

          // extend 
          if (level == 0) {
            graph_node_t cur_job, njobs;

            // TODO: change to warp
            if (threadIdx.x % WARP_SIZE == 0) {
              get_job(q, cur_job, njobs);
              stk->iter[0] = 0;
              for (size_t i = 0; i < njobs; i++) {
                stk->slot_storage[0][0][i] = q->queue[cur_job + i];
              }
              stk->slot_size[0][0] = njobs;
            }
            __syncwarp();
          }
          else {

            for (pattern_node_t i = 0; i < PAT_SIZE; i++) {

              if (pat->set_ops[level][i] < 0) break;

              pattern_node_t slot_idx = (pat->set_ops[level][i] & 0xF);

              // TODO: compute ub based on pattern->partial
              graph_node_t ub = INT_MAX;

              graph_node_t* neighbor = g->rowptr[stk->path[level]] + g->colidx;
              graph_node_t neighbor_size = (graph_node_t)(g->rowptr[stk->path[level] + 1] - g->rowptr[stk->path[level]]);

              if (pat->set_ops[level][i] & 0x10) {
                // TODO: change to warp 
                for (graph_node_t j = 0; j < neighbor_size; j++) {
                  stk->slot_storage[level][i][j] = neighbor[j];
                }
                stk->slot_size[level][i] = neighbor_size;

                for (pattern_node_t j = level - 1; j >= 0; j--) {
                  graph_node_t* nb = g->rowptr[stk->path[j]] + g->colidx;
                  graph_node_t nsize = (graph_node_t)(g->rowptr[stk->path[j] + 1] - g->rowptr[stk->path[j]]);

                  difference(&(stk->slot_storage[level][i][0]), nb, stk->slot_size[level][i], nsize, ub);
                }

              }
              else {
                // TODO: change to warp
                for (graph_node_t j = 0; j < stk->slot_size[level - 1][slot_idx]; j++) {
                  stk->slot_storage[level][i][j] = stk->slot_storage[level - 1][slot_idx][j];
                }
                stk->slot_size[level][i] = stk->slot_size[level - 1][slot_idx];

                if (pat->set_ops[level][i] & 0x20) {
                  difference(&(stk->slot_storage[level][i][0]), neighbor, stk->slot_size[level][i], neighbor_size, ub);
                }
                else {
                  intersection(&(stk->slot_storage[level][i][0]), neighbor, stk->slot_size[level][i], neighbor_size, ub);
                }
              }
            }
          }

          stk->iter[level] = 0;
        }
        if (stk->iter[level] < stk->slot_size[level][0]) {
          stk->path[level] = stk->slot_storage[level][0][stk->iter[level]];
          level++;
        }
        else {
          stk->slot_size[level][0] = 0;
          level--;
          if (level < 0) {
            break;
          }
          if (threadIdx.x % WARP_SIZE == 0) stk->iter[level]++;
          __syncwarp();
        }
      }
      else if (level == pat->nnodes - 1) {
        // TODO: we can save the storage of sets for the last level
        if (threadIdx.x % WARP_SIZE == 0) {
          count += stk->slot_size[level][0];
        }
        __syncwarp();
        stk->slot_size[level][0] = 0;
        level--;
      }
    }
  }


  __global__ void _parallel_match(Graph* dev_graph, Pattern* dev_pattern, CallStack* dev_callstack, JobQueue* job_queue) {

    __shared__ Graph graph;
    __shared__ Pattern pat;
    __shared__ JobQueue queue;
    __shared__ CallStack stk[NWARPS_PER_BLOCK];
    __shared__ size_t count[NWARPS_PER_BLOCK];

    int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
    int global_wid = global_tid / WARP_SIZE;
    int local_wid = threadIdx.x / WARP_SIZE;

    // TODO: change to warp 
    // graph and pattern are shared among all threads in a threadblock
    if (threadIdx.x == 0) {
      graph = *dev_graph;
      pat = *dev_pattern;
      queue = *job_queue;
    }
    __syncwarp();

    // TODO: change to warp
    // callstack is shared among all threads in each warp
    if (threadIdx.x % WARP_SIZE == 0) {
      stk[local_wid] = dev_callstack[global_wid];
    }
    __syncwarp();

    while (true) {
      match(&graph, &pat, &stk[local_wid], &queue, &count[local_wid]);
      break;
      // TODO: load balance
    }
  }
}