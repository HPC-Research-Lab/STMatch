#include "hip/hip_runtime.h"
#include "KernelLauncher.cuh"
#include <stdio.h>

namespace libra {

#ifdef __HIPCC__

  __global__ void _parallel_match(Graph* dev_graph, Pattern* dev_pattern, CallStack* dev_callstack, JobQueue* job_queue) {

    extern __shared__ char graph_pat_pm_mem[];

    // cache the graph, pattern, and callstack on shared memory
    Graph* graph = (Graph*)graph_pat_pm_mem;
    Pattern* pat = (Pattern*)(graph_pat_pm_mem + sizeof(Graph));
    PatternMatcher* pm = (PatternMatcher*)(graph_pat_pm_mem + sizeof(Pattern) + sizeof(Graph));

    int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
    int global_wid = global_tid / WARP_SIZE;
    int local_wid = threadIdx.x / WARP_SIZE;

    // TODO: change this to multiple threads
    // graph and pattern are shared among all threads in a threadblock
    if (threadIdx.x == 0) {
      *graph = *dev_graph;
      *pat = *dev_pattern;
    }


    // TODO: change this to multiple threads
    // callstack is shared among all threads in each warp
    if (threadIdx.x % WARP_SIZE == 0) {
      pm[local_wid].pattern_ = pat;
      pm[local_wid].graph_ = graph;
      pm[local_wid].stk_ = dev_callstack[global_wid];
      pm[local_wid].job_queue_ = job_queue;
    }




    while (true) {
      pm[local_wid].match();
      break;
      // TODO: load balance
    }
  }

  __device__ void lower_bound(size_t* nums, size_t& nums_size, size_t target) {
    ssize_t mid;
    ssize_t low = 0;
    ssize_t high = nums_size;

    while (low < high) {
      mid = low + (high - low) / 2;

      if (target <= nums[mid]) {
        high = mid;
      }
      else {
        low = mid + 1;
      }
    }
    if (low < nums_size && nums[low] < target) {
      low++;
    }
    nums_size = low;
  }


  __device__ bool _lower_bound_exist(size_t* _arr, size_t _size, size_t _target) {
    ssize_t mid;
    ssize_t low = 0;
    ssize_t high = _size;
    while (low < high) {
      mid = low + (high - low) / 2;
      if (_target <= _arr[mid]) {
        high = mid;
      }
      else {
        low = mid + 1;
      }
    }
    if (low < _size && _arr[low] < _target) {
      low++;
    }
    return (low < _size&& _arr[low] == _target);
  }


  __device__ void _prefix_sum(int* _input) {

    int thid = threadIdx.x;
    int offset = 1;
    for (int d = warpSize >> 1; d > 0; d >>= 1) // build sum in place up the tree
    {
      //__syncthreads();
      if (thid < d)
      {
        int ai = offset * (2 * thid + 1) - 1;
        int bi = offset * (2 * thid + 2) - 1;
        _input[bi] += _input[ai];
      }
      offset *= 2;
    }
    if (thid == 0) { _input[warpSize - 1] = 0; } // clear the last element
    for (int d = 1; d < warpSize; d *= 2) // traverse down tree & build scan
    {
      offset >>= 1;
      //__syncthreads();
      if (thid < d)
      {
        int ai = offset * (2 * thid + 1) - 1;
        int bi = offset * (2 * thid + 2) - 1;
        int t = _input[ai];
        _input[ai] = _input[bi];
        _input[bi] += t;
      }
    }
    //__syncthreads();
  }

  __device__ size_t intersection(size_t* _first, size_t* _second, size_t _first_size, size_t _second_size, size_t* _res, size_t _up) {
    __shared__ int pos[32];
    int tid = threadIdx.x;
    int end_pos = 0;
    int loop_end = _first_size - _first_size % warpSize + warpSize;
    bool last_find;
    for (int idx = tid; idx < loop_end && _first[idx] < _up; idx += warpSize) {
      last_find = false;
      pos[tid] = 0;
      if (idx < _first_size && _first[idx] < _up) {
        if (_lower_bound_exist(_second, _second_size, _first[idx])) {
          pos[tid] = 1;
        }
        if (threadIdx.x == warpSize - 1 && pos[warpSize - 1] == 1) {
          last_find = true;
        }
      }
      _prefix_sum(pos);

      if ((tid == warpSize - 1 && last_find) ||
        tid != warpSize - 1 && pos[tid] < pos[tid + 1]) {
        _res[end_pos + pos[tid]] = _first[idx];
      }
      end_pos += pos[warpSize - 1];
    }
    return end_pos;
  }

  __device__ size_t difference(size_t* _first, size_t* _second, size_t _first_size, size_t _second_size, size_t* _res, size_t _up) {
    __shared__ int pos[32];
    int tid = threadIdx.x;
    int end_pos = 0;
    int loop_end = _first_size - _first_size % warpSize + warpSize;
    bool last_find;
    for (int idx = tid; idx < loop_end; idx += warpSize) {
      last_find = false;
      pos[tid] = 0;

      if (idx < _first_size && _first[idx] < _up) {
        if (!_lower_bound_exist(_second, _second_size, _first[idx])) {
          pos[tid] = 1;
        }
        if (threadIdx.x == warpSize - 1 && pos[warpSize - 1] == 1) {
          last_find = true;
        }
      }
      _prefix_sum(pos);

      if ((tid == warpSize - 1 && last_find) ||
        tid != warpSize - 1 && pos[tid] < pos[tid + 1]) {
        _res[end_pos + pos[tid]] = _first[idx];
      }
      end_pos += pos[warpSize - 1];
    }
    return end_pos;
  }

#else

  size_t intersection(size_t* first, size_t* second, size_t first_size, size_t second_size, size_t* res, size_t up) {
    size_t pi = 0, pj = 0, pos = 0;
    while (pi != first_size && pj != second_size) {
      if (first[pi] >= up) break;
      if (second[pj] >= up) break;
      if (first[pi] < second[pj])
        pi++;
      else if (first[pi] > second[pj])
        pj++;
      else {
        res[pos++] = first[pi];
        pi++;
        pj++;
      }
    }
    return pos;
  }


  inline int difference(size_t* first, size_t* second, int first_size, int second_size, size_t* res, size_t up) {
    int pi = 0, pj = 0, pos = 0;
    while (pi != first_size && pj != second_size) {
      int left = first[pi]; int right = second[pj];
      if (left >= up) break;
      if (right >= up) break;
      if (left <= right) pi++;
      if (right <= left) pj++;
      if (left < right) {
        res[pos++] = left;
      }
    }
    while (pi < first_size) {
      int left = first[pi++];
      if (left >= up) break;
      res[pos++] = left;
    }
    return pos;
  }

  int difference_ns(size_t* first, size_t* second, int first_size, int second_size, size_t* res, size_t up) {
    int pi = 0, pj = 0, pos = 0;
    while (pi != first_size && pj != second_size) {
      if (first[pi] >= up) break;
      if (second[pj] >= up) break;
      if (first[pi] < second[pj]) {
        //res[pos] = first[pi];
        pi++;
        pos++;
      }
      else {
        if (first[pi] == second[pj]) {
          ++pi;
        }
        ++pj;
      }
    }
    for (int i = pi; i < first_size; i++) {
      if (first[i] >= up) break;
      pos++;
      //res[pos++]=first[i];
    }
    return pos;

    return pos;
  }

  size_t lower_bound(size_t* nums, size_t nums_size, size_t target) {
    size_t mid;
    size_t low = 0;
    size_t high = nums_size;

    while (low < high) {
      mid = low + (high - low) / 2;

      if (target <= nums[mid]) {
        high = mid;
      }
      else {
        low = mid + 1;
      }
    }
    if (low < nums_size && nums[low] < target) {
      low++;
    }
    return low;
  }

#endif

}