#include "hip/hip_runtime.h"
#include "gpu_match.cuh"
#include <hip/hip_runtime.h>

namespace libra {

  // TODO: implement intersection and difference

  template<typename DATA_T, typename SIZE_T>
  __device__
    inline bool lower_bound_exist(DATA_T* set2, SIZE_T set2_size, DATA_T target) {
    int mid;
    int low = 0;
    int high = set2_size - 1;
    while (low <= high) {
      mid = (low + high) / 2;
      if (target == set2[mid]) {
        return true;
      }
      else if (target > set2[mid]) {
        low = mid + 1;
      }
      else {
        high = mid - 1;
      }
    }
    return false;
  }


  __device__ inline void prefix_sum(int* _input) {

    int thid = threadIdx.x % WARP_SIZE;
    int offset = 1;
    int last_element = _input[WARP_SIZE - 1];
    // build sum in place up the tree
    for (int d = WARP_SIZE >> 1; d > 0; d >>= 1) {
      //__syncthreads();
      if (thid < d) {
        int ai = offset * (2 * thid + 1) - 1;
        int bi = offset * (2 * thid + 2) - 1;
        _input[bi] += _input[ai];
      }
      offset <<= 1;
    }
    if (thid == 0) { _input[WARP_SIZE - 1] = 0; } // clear the last element
     // traverse down tree & build scan
    for (int d = 1; d < WARP_SIZE; d <<= 1) {
      offset >>= 1;
      //__syncthreads();
      if (thid < d) {
        int ai = offset * (2 * thid + 1) - 1;
        int bi = offset * (2 * thid + 2) - 1;
        int t = _input[ai];
        _input[ai] = _input[bi];
        _input[bi] += t;
      }
    }

    if (thid == WARP_SIZE - 1)
      _input[WARP_SIZE] = _input[WARP_SIZE - 1] + last_element;
  }


  template<typename DATA_T, typename SIZE_T>
  __device__ void intersection(DATA_T* set1, DATA_T* set2, DATA_T* _res, SIZE_T set1_size, SIZE_T set2_size, SIZE_T* res_size, DATA_T ub) {

    __shared__ int pos[NWARPS_PER_BLOCK][33];
    __shared__ bool still_loop[NWARPS_PER_BLOCK];

    int wid = threadIdx.x / WARP_SIZE;
    int tid = threadIdx.x % WARP_SIZE;

    int end_pos = 0;

    still_loop[wid] = true;

    for (int idx = tid; (idx < (((set1_size - 1) / WARP_SIZE + 1) * WARP_SIZE) && still_loop[wid]); idx += WARP_SIZE) {
      pos[wid][tid] = 0;
      pos[wid][WARP_SIZE] = 0;
      if (idx < set1_size && set1[idx] < ub) {
        if (lower_bound_exist(set2, set2_size, set1[idx])) {
          pos[wid][tid] = 1;
        }
      }
      else {
        still_loop[wid] = false;
      }
      __syncwarp();

      prefix_sum(&pos[wid][0]);

      if (pos[wid][tid + 1] > pos[wid][tid]) {
        _res[end_pos + pos[wid][tid]] = set1[idx];
      }
      end_pos += pos[wid][WARP_SIZE];
    }

    *res_size = end_pos;
  }

  template<typename DATA_T, typename SIZE_T>
  __device__ void difference(DATA_T* set1, DATA_T* set2, DATA_T* _res, SIZE_T set1_size, SIZE_T set2_size, SIZE_T* res_size, DATA_T ub) {
    __shared__ int pos[NWARPS_PER_BLOCK][33];
    __shared__ bool still_loop[NWARPS_PER_BLOCK];

    int wid = threadIdx.x / WARP_SIZE;
    int tid = threadIdx.x % WARP_SIZE;

    int end_pos = 0;

    still_loop[wid] = true;

    for (int idx = tid; (idx < (((set1_size - 1) / WARP_SIZE + 1) * WARP_SIZE) && still_loop[wid]); idx += WARP_SIZE) {
      pos[wid][tid] = 0;
      pos[wid][WARP_SIZE] = 0;
      if (idx < set1_size && set1[idx] < ub) {
        if (set2 != NULL) {
          if (!lower_bound_exist(set2, set2_size, set1[idx])) {
            pos[wid][tid] = 1;
          }
        }
        else {
          pos[wid][tid] = 1;
        }
      }
      else {
        still_loop[wid] = false;
      }
      __syncwarp();

      prefix_sum(&pos[wid][0]);

      // TODO: if _res and set1 are the same
      if (pos[wid][tid + 1] > pos[wid][tid]) {
        _res[end_pos + pos[wid][tid]] = set1[idx];
      }
      end_pos += pos[wid][WARP_SIZE];
    }
    *res_size = end_pos;
  }

  __device__ void lock(int* mutex) {
    while (atomicCAS(mutex, 0, 1) != 0);
  }

  __device__ void unlock(int* mutex) {
    atomicExch(mutex, 0);
  }

  __device__ void get_job(JobQueue* q, graph_node_t& cur_pos, graph_node_t& njobs) {
    lock(&(q->mutex));
    cur_pos = q->cur;
    q->cur += JOB_CHUNK_SIZE;
    if (q->cur > q->length) q->cur = q->length;
    njobs = q->cur - cur_pos;
    unlock(&(q->mutex));
  }

  __device__ void extend(Graph* g, Pattern* pat, CallStack* stk, JobQueue* q, graph_node_t level) {
    if (level == 0) {
      graph_node_t cur_job, njobs;

      // TODO: change to warp
      if (threadIdx.x % WARP_SIZE == 0) {
        get_job(q, cur_job, njobs);
        // printf("%d %d: %d %d\n", blockIdx.x, threadIdx.x, cur_job, njobs);
        for (size_t i = 0; i < njobs; i++) {
          stk->slot_storage[0][0][i] = q->queue[cur_job + i];
        }
        stk->slot_size[0][0] = njobs;
      }
      __syncwarp();
    }
    else {

      for (pattern_node_t i = 0; i < PAT_SIZE; i++) {

        // compute ub based on pattern->partial
        graph_node_t ub = INT_MAX;
        if (pat->partial[level - 1][i] >= 0) ub = stk->path[pat->partial[level - 1][i]];
        graph_node_t* neighbor = &g->colidx[g->rowptr[stk->path[level - 1]]];
        graph_node_t neighbor_size = (graph_node_t)(g->rowptr[stk->path[level - 1] + 1] - g->rowptr[stk->path[level - 1]]);

        if (pat->set_ops[level - 1][i] & 0x10) {

          graph_node_t* nb = NULL;
          graph_node_t nsize = 0;

          if (level >= 2) {
            nb = &g->colidx[g->rowptr[stk->path[level - 2]]];
            nsize = (graph_node_t)(g->rowptr[stk->path[level - 2] + 1] - g->rowptr[stk->path[level - 2]]);
          }
          // when the second set is empty, the difference function simply checks ub and copy first set to res set.
          difference(neighbor, nb, &(stk->slot_storage[level][i][0]), neighbor_size, nsize, &(stk->slot_size[level][i]), ub);

          for (pattern_node_t j = level - 3; j >= 0; j--) {
            nb = &g->colidx[g->rowptr[stk->path[j]]];
            nsize = (graph_node_t)(g->rowptr[stk->path[j] + 1] - g->rowptr[stk->path[j]]);

            difference(&(stk->slot_storage[level][i][0]), nb, &(stk->slot_storage[level][i][0]), stk->slot_size[level][i], nsize, &(stk->slot_size[level][i]), ub);
          }
        }
        else {

          pattern_node_t slot_idx = (pat->set_ops[level - 1][i] & 0xF);

          if (pat->set_ops[level - 1][i] & 0x20) {
            intersection(&(stk->slot_storage[level - 1][slot_idx][0]), neighbor, &(stk->slot_storage[level][i][i]), stk->slot_size[level - 1][slot_idx], neighbor_size, &(stk->slot_size[level][i]), ub);
          }
          else {
            difference(&(stk->slot_storage[level - 1][slot_idx][0]), neighbor, &(stk->slot_storage[level][i][i]), stk->slot_size[level - 1][slot_idx], neighbor_size, &(stk->slot_size[level][i]), ub);
          }
        }

        if (pat->set_ops[level - 1][i] < 0) break;
      }
    }
    stk->iter[level] = 0;
  }

  __device__ void match(Graph* g, Pattern* pat, CallStack* stk, JobQueue* q, size_t* count) {
    graph_node_t level = 0;

    while (true) {

      if (level < pat->nnodes - 1) {
        if (stk->slot_size[level][0] == 0) {

          extend(g, pat, stk, q, level);
          if (level == 0 && stk->slot_size[level][0] == 0) break;
        }
        if (stk->iter[level] < stk->slot_size[level][0]) {
          stk->path[level] = stk->slot_storage[level][0][stk->iter[level]];
          level++;
        }
        else {
          stk->slot_size[level][0] = 0;
          if (level > 0) {
            level--;
            if (threadIdx.x % WARP_SIZE == 0) stk->iter[level]++;
            __syncwarp();
          }
        }
      }
      else if (level == pat->nnodes - 1) {

        // TODO: we can save the storage of sets for the last level
        extend(g, pat, stk, q, level);
        if (threadIdx.x % WARP_SIZE == 0) {
          *count += stk->slot_size[level][0];
        }
        __syncwarp();
        stk->slot_size[level][0] = 0;
        level--;
        if (threadIdx.x % WARP_SIZE == 0) stk->iter[level]++;
        __syncwarp();
      }
    }
  }

  __global__ void _parallel_match(Graph* dev_graph, Pattern* dev_pattern, CallStack* dev_callstack, JobQueue* job_queue, size_t* res) {

    __shared__ Graph graph;
    __shared__ Pattern pat;
    __shared__ CallStack stk[NWARPS_PER_BLOCK];
    __shared__ size_t count[NWARPS_PER_BLOCK];

    int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
    int global_wid = global_tid / WARP_SIZE;
    int local_wid = threadIdx.x / WARP_SIZE;

    if (threadIdx.x == 0) {
      graph = *dev_graph;
      pat = *dev_pattern;
    }
    __syncthreads();

    if (threadIdx.x % WARP_SIZE == 0) {
      stk[local_wid] = dev_callstack[global_wid];
    }
    __syncwarp();

    auto start = clock64();
    while (true) {
      match(&graph, &pat, &stk[local_wid], job_queue, &count[local_wid]);
      break;
      // TODO: load balance
    }
    auto stop = clock64();

    if (threadIdx.x % WARP_SIZE == 0) {
      res[global_wid] = count[local_wid];
      // printf("%ld\n", stop - start);
    }
  }
}