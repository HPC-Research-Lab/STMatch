#include "hip/hip_runtime.h"
#include "gpu_match.cuh"
#include <hip/hip_runtime.h>

// TODO: change this to gpu array so we can use different unroll for diffrent levels
#define UNROLL_SIZE(l) UNROLL

namespace libra {

  inline __device__ graph_node_t path(CallStack* stk, Pattern* pat, int level, int k) {
    return stk->slot_storage[pat->rowptr[level]][stk->uiter[level]][stk->iter[level] + k];
  }

  typedef struct {
    graph_node_t* set1[UNROLL], * set2[UNROLL], * res[UNROLL];
    graph_node_t set1_size[UNROLL], set2_size[UNROLL], * res_size[UNROLL];
    graph_node_t ub[UNROLL];
    bitarray32 label;
    Graph* g;
    int num_sets;
  } Arg_t;

  template<typename DATA_T, typename SIZE_T>
  __device__
    inline bool bsearch_exist(DATA_T* set2, SIZE_T set2_size, DATA_T target) {
    int mid;
    int low = 0;
    int high = set2_size - 1;
    while (low <= high) {
      mid = (low + high) / 2;
      if (target == set2[mid]) {
        return true;
      }
      else if (target > set2[mid]) {
        low = mid + 1;
      }
      else {
        high = mid - 1;
      }
    }
    return false;
  }

  template<typename DATA_T, typename SIZE_T>
  inline __device__
    SIZE_T upper_bound(DATA_T* set2, SIZE_T set2_size, DATA_T target) {
    int i, step;
    int low = 0;
    while (set2_size > 0) {
      i = low;
      step = set2_size / 2;
      i += step;
      if (target > set2[i]) {
        low = ++i; set2_size -= step + 1;
      }
      else {
        set2_size = step;
      }
    }
    return low;
  }

  inline __device__
    void prefix_sum(int* _input, int input_size) {

    int thid = threadIdx.x % WARP_SIZE;
    int offset = 1;
    int last_element = _input[input_size - 1];
    // build sum in place up the tree
    for (int d = (WARP_SIZE >> 1); d > 0; d >>= 1) {
      if (thid < d) {
        int ai = offset * (2 * thid + 1) - 1;
        int bi = offset * (2 * thid + 2) - 1;
        _input[bi] += _input[ai];
      }
      offset <<= 1;
    }
    if (thid == 0) { _input[WARP_SIZE - 1] = 0; } // clear the last element
     // traverse down tree & build scan
    for (int d = 1; d < WARP_SIZE; d <<= 1) {
      offset >>= 1;
      if (thid < d) {
        int ai = offset * (2 * thid + 1) - 1;
        int bi = offset * (2 * thid + 2) - 1;
        int t = _input[ai];
        _input[ai] = _input[bi];
        _input[bi] += t;
      }
    }
    __syncwarp();

    if (thid >= input_size - 1)
      _input[thid + 1] = _input[input_size - 1] + last_element;
  }


  template<bool DIFF>
  __device__ void compute_set(Arg_t* arg) {

    __shared__ int pos[NWARPS_PER_BLOCK][WARP_SIZE + 1];
    __shared__ graph_node_t size_psum[NWARPS_PER_BLOCK][WARP_SIZE + 1];
    __shared__ int end_pos[NWARPS_PER_BLOCK][UNROLL];

    int wid = threadIdx.x / WARP_SIZE;
    int tid = threadIdx.x % WARP_SIZE;


    if (tid < arg->num_sets) {
      arg->set1_size[tid] = upper_bound(arg->set1[tid], arg->set1_size[tid], arg->ub[tid]);
      size_psum[wid][tid] = arg->set1_size[tid];
      end_pos[wid][tid] = 0;
    }
    else {
      size_psum[wid][tid] = 0;
    }
    __syncwarp();

    prefix_sum(&size_psum[wid][0], arg->num_sets);
    __syncwarp();


    bool still_loop = true;
    int slot_idx = 0;
    int offset = 0;

    int size1 = (size_psum[wid][WARP_SIZE] > 0) ? (((size_psum[wid][WARP_SIZE] - 1) / WARP_SIZE + 1) * WARP_SIZE) : 0;

    for (int idx = tid; (idx < size1 && still_loop); idx += WARP_SIZE) {
      pos[wid][tid] = 0;

      if (idx < size_psum[wid][WARP_SIZE]) {

        while (idx >= size_psum[wid][slot_idx + 1]) {
          slot_idx++;
        }
        offset = idx - size_psum[wid][slot_idx];

        bitarray32 lb = arg->g->vertex_label[arg->set1[slot_idx][offset]];
        if ((lb && arg->label == lb) && (DIFF ^ bsearch_exist(arg->set2[slot_idx], arg->set2_size[slot_idx], arg->set1[slot_idx][offset]))) {
          pos[wid][tid] = 1;
        }
      }
      else {
        slot_idx = arg->num_sets;
        still_loop = false;
      }

      still_loop = __shfl_sync(0xFFFFFFFF, still_loop, 31);

      prefix_sum(&pos[wid][0], WARP_SIZE);
      __syncwarp();

      graph_node_t res_tmp;
      if (pos[wid][tid + 1] > pos[wid][tid]) {
        res_tmp = arg->set1[slot_idx][offset];
      }
      __syncwarp();
      int prev_idx = ((idx / WARP_SIZE == size_psum[wid][slot_idx] / WARP_SIZE) ? size_psum[wid][slot_idx] % WARP_SIZE : 0);

      if (pos[wid][tid + 1] > pos[wid][tid]) {
        arg->res[slot_idx][end_pos[wid][slot_idx] + pos[wid][tid] - pos[wid][prev_idx]] = res_tmp;
      }

      if (slot_idx < __shfl_down_sync(0xFFFFFFFF, slot_idx, 1)) {
        end_pos[wid][slot_idx] += pos[wid][tid + 1] - pos[wid][prev_idx];
      }
      else if (tid == WARP_SIZE - 1 && slot_idx < arg->num_sets) {
        end_pos[wid][slot_idx] += pos[wid][WARP_SIZE] - pos[wid][prev_idx];
      }
    }
    __syncwarp();
    if (tid < arg->num_sets) {
      *(arg->res_size[tid]) = end_pos[wid][tid];
    }
    __syncwarp();
  }

  __device__ void lock(int* mutex) {
    while (atomicCAS(mutex, 0, 1) != 0);
  }

  __device__ void unlock(int* mutex) {
    atomicExch(mutex, 0);
  }

  __device__ void get_job(JobQueue* q, graph_node_t& cur_pos, graph_node_t& njobs) {
    lock(&(q->mutex));
    cur_pos = q->cur;
    q->cur += JOB_CHUNK_SIZE;
    if (q->cur > q->length) q->cur = q->length;
    njobs = q->cur - cur_pos;
    unlock(&(q->mutex));
  }

  __device__ void extend(Graph* g, Pattern* pat, CallStack* stk, JobQueue* q, pattern_node_t level) {

    __shared__ Arg_t arg[NWARPS_PER_BLOCK];
    __shared__ graph_node_t cur_job_njobs[NWARPS_PER_BLOCK][2];
    int wid = threadIdx.x / WARP_SIZE;

    if (level == 0) {

      int tid = threadIdx.x % WARP_SIZE;

      for (int k = 0; k < UNROLL_SIZE(level); k++) {
        if (threadIdx.x % WARP_SIZE == 0) {
          get_job(q, cur_job_njobs[wid][0], cur_job_njobs[wid][1]);
        }
        __syncwarp();
        if (tid < cur_job_njobs[wid][1]) {
          for (int j = 0; j < q->start_level; j++) {
            stk->slot_storage[0][k][tid + JOB_CHUNK_SIZE * j] = (q->q[cur_job_njobs[wid][0] + tid].nodes)[j];
          }
        }
        stk->slot_size[0][k] = cur_job_njobs[wid][1];
        stk->start_level = q->start_level;
      }
      __syncwarp();
    }
    else {

    arg[wid].g = g;
    arg[wid].num_sets = UNROLL_SIZE(level);

    int remaining = stk->slot_size[pat->rowptr[level - 1]][stk->uiter[level - 1]] - stk->iter[level - 1];
    if (remaining >= 0 && UNROLL_SIZE(level) > remaining) {
      arg[wid].num_sets = remaining;
    }

    for (int i = pat->rowptr[level]; i < pat->rowptr[level + 1]; i++) {

      // compute ub based on pattern->partial
      graph_node_t ub = INT_MAX;
      // assert(level >= 1);
      if (i == pat->rowptr[level]) {
        ub = INT_MAX;
        if (pat->partial[i] != 0) {
          for (pattern_node_t k = 0; k < level - 1; k++) {
            if ((pat->partial[i] & (1 << k)) && (ub > path(stk, pat, k, stk->uiter[k + 1]))) ub = path(stk, pat, k, stk->uiter[k + 1]);
          }

          for (pattern_node_t k = 0; k < arg[wid].num_sets; k++) {
            arg[wid].ub[k] = ub;
            if ((pat->partial[i] & (1 << (level - 1))) && (arg[wid].ub[k] > path(stk, pat, level - 1, k))) arg[wid].ub[k] = path(stk, pat, level - 1, k);
          }
        }
        else {
          for (pattern_node_t k = 0; k < arg[wid].num_sets; k++) {
            arg[wid].ub[k] = INT_MAX;
          }
        }
      }
      else {
        ub = -1;
        if (pat->partial[i] != 0) {
          for (pattern_node_t k = 0; k < level - 1; k++) {
            if ((pat->partial[i] & (1 << k)) && (ub < path(stk, pat, k, stk->uiter[k + 1]))) ub = path(stk, pat, k, stk->uiter[k + 1]);
          }
          for (pattern_node_t k = 0; k < arg[wid].num_sets; k++) {
            arg[wid].ub[k] = ub;
            if ((pat->partial[i] & (1 << (level - 1))) && (arg[wid].ub[k] < path(stk, pat, level - 1, k))) arg[wid].ub[k] = path(stk, pat, level - 1, k);
            if (arg[wid].ub[k] == -1) arg[wid].ub[k] = INT_MAX;
          }
        }
        else {
          for (pattern_node_t k = 0; k < arg[wid].num_sets; k++) {
            arg[wid].ub[k] = INT_MAX;
          }
        }
      }

      arg[wid].label = pat->slot_labels[i];

      if (pat->set_ops[i] & 0x20) {

        graph_node_t* nb = NULL;
        graph_node_t nsize = 0;

        if (!EDGE_INDUCED) {
          if (level >= 2) {
            nb = &g->colidx[g->rowptr[path(stk, pat, level - 2, stk->uiter[level - 1])]];
            nsize = (graph_node_t)(g->rowptr[path(stk, pat, level - 2, stk->uiter[level - 1]) + 1] - g->rowptr[path(stk, pat, level - 2, stk->uiter[level - 1])]);
          }
        }
        for (graph_node_t k = 0; k < arg[wid].num_sets; k++) {
          graph_node_t* neighbor = &g->colidx[g->rowptr[path(stk, pat, level - 1, k)]];
          graph_node_t neighbor_size = (graph_node_t)(g->rowptr[path(stk, pat, level - 1, k) + 1] - g->rowptr[path(stk, pat, level - 1, k)]);
          arg[wid].set1[k] = neighbor;
          arg[wid].set2[k] = nb;
          arg[wid].res[k] = &(stk->slot_storage[i][k][0]);
          arg[wid].set1_size[k] = neighbor_size;
          arg[wid].set2_size[k] = nsize;
          arg[wid].res_size[k] = &(stk->slot_size[i][k]);
        }
        compute_set<true>(&arg[wid]);
        for (graph_node_t k = arg[wid].num_sets; k < UNROLL_SIZE(level); k++) stk->slot_size[i][k] = 0;


        if (!EDGE_INDUCED) {
          for (pattern_node_t j = level - 3; j >= 0; j--) {

            nb = &g->colidx[g->rowptr[path(stk, pat, j, stk->uiter[j + 1])]];
            nsize = (graph_node_t)(g->rowptr[path(stk, pat, j, stk->uiter[j + 1]) + 1] - g->rowptr[path(stk, pat, j, stk->uiter[j + 1])]);

            for (graph_node_t k = 0; k < arg[wid].num_sets; k++) {
              arg[wid].set1[k] = &(stk->slot_storage[i][k][0]);
              arg[wid].set2[k] = nb;
              arg[wid].res[k] = &(stk->slot_storage[i][k][0]);
              arg[wid].set1_size[k] = stk->slot_size[i][k];
              arg[wid].set2_size[k] = nsize;
              arg[wid].res_size[k] = &(stk->slot_size[i][k]);
            }
            compute_set<true>(&arg[wid]);
            for (graph_node_t k = arg[wid].num_sets; k < UNROLL_SIZE(level); k++) stk->slot_size[i][k] = 0;
          }
        }
      }
      else {

        pattern_node_t slot_idx = (pat->set_ops[i] & 0x1F);

        if (pat->set_ops[i] & 0x40) {
          for (graph_node_t k = 0; k < arg[wid].num_sets; k++) {
            graph_node_t* neighbor = &g->colidx[g->rowptr[path(stk, pat, level - 1, k)]];
            graph_node_t neighbor_size = (graph_node_t)(g->rowptr[path(stk, pat, level - 1, k) + 1] - g->rowptr[path(stk, pat, level - 1, k)]);

            arg[wid].set1[k] = &(stk->slot_storage[slot_idx][stk->uiter[level - 1]][0]);
            arg[wid].set2[k] = neighbor;
            arg[wid].res[k] = &(stk->slot_storage[i][k][0]);
            arg[wid].set1_size[k] = stk->slot_size[slot_idx][stk->uiter[level - 1]];
            arg[wid].set2_size[k] = neighbor_size;
            arg[wid].res_size[k] = &(stk->slot_size[i][k]);
          }
          compute_set<false>(&arg[wid]);
          for (graph_node_t k = arg[wid].num_sets; k < UNROLL_SIZE(level); k++) stk->slot_size[i][k] = 0;

        }
        else {

          for (graph_node_t k = 0; k < arg[wid].num_sets; k++) {
            graph_node_t* neighbor = NULL;
            graph_node_t neighbor_size = 0;
            if (!EDGE_INDUCED) {
              neighbor = &g->colidx[g->rowptr[path(stk, pat, level - 1, k)]];
              neighbor_size = (graph_node_t)(g->rowptr[path(stk, pat, level - 1, k) + 1] - g->rowptr[path(stk, pat, level - 1, k)]);
            }
            arg[wid].set1[k] = &(stk->slot_storage[slot_idx][stk->uiter[level - 1]][0]);
            arg[wid].set2[k] = neighbor;
            arg[wid].res[k] = &(stk->slot_storage[i][k][0]);
            arg[wid].set1_size[k] = stk->slot_size[slot_idx][stk->uiter[level - 1]];
            arg[wid].set2_size[k] = neighbor_size;
            arg[wid].res_size[k] = &(stk->slot_size[i][k]);
          }
          compute_set<true>(&arg[wid]);
          for (graph_node_t k = arg[wid].num_sets; k < UNROLL_SIZE(level); k++) stk->slot_size[i][k] = 0;

        }
      }
    }
    }
    stk->iter[level] = 0;
    stk->uiter[level] = 0;
}

__device__ void match(Graph* g, Pattern* pat, CallStack* stk, JobQueue* q, size_t* count) {
  pattern_node_t level = 0;

  while (true) {

    if (level < pat->nnodes - 1) {

      if (stk->uiter[level] == 0 && stk->slot_size[pat->rowptr[level]][0] == 0) {
        extend(g, pat, stk, q, level);
        if (level == 0 && stk->slot_size[0][0] == 0) break;
      }

      if (stk->uiter[level] < UNROLL_SIZE(level)) {
        if (stk->iter[level] < stk->slot_size[pat->rowptr[level]][stk->uiter[level]]) {
          if (level > 0 && level < stk->start_level && path(stk, pat, level, 0) != stk->slot_storage[0][stk->uiter[0]][stk->iter[0] + stk->uiter[1] + level * JOB_CHUNK_SIZE]) {
            stk->iter[level]++;
          }
          else if (level > 1 && level < stk->start_level + 1 && path(stk, pat, level - 1, stk->uiter[level]) != stk->slot_storage[0][stk->uiter[0]][stk->iter[0] + stk->uiter[1] + (level - 1) * JOB_CHUNK_SIZE]) {
            stk->uiter[level]++;
          }
          else {
            level++;
          }
        }
        else {
          stk->slot_size[pat->rowptr[level]][stk->uiter[level]] = 0;
          stk->iter[level] = 0;
          stk->uiter[level]++;
        }
      }
      else {
        stk->uiter[level] = 0;
        if (level > 0) {
          level--;
          if (threadIdx.x % WARP_SIZE == 0) stk->iter[level] += UNROLL_SIZE(level + 1);
          __syncwarp();
        }
      }
    }
    else if (level == pat->nnodes - 1) {

      // TODO: we can save the storage of sets for the last level
      extend(g, pat, stk, q, level);
      for (int j = 0; j < UNROLL_SIZE(level); j++) {
        if (threadIdx.x % WARP_SIZE == 0) {
          *count += stk->slot_size[pat->rowptr[level]][j];
        }
        __syncwarp();
        stk->slot_size[pat->rowptr[level]][j] = 0;
      }
      level--;
      if (threadIdx.x % WARP_SIZE == 0) stk->iter[level] += UNROLL_SIZE(level + 1);
      __syncwarp();
    }
  }
}

__global__ void _parallel_match(Graph* dev_graph, Pattern* dev_pattern, CallStack* dev_callstack, JobQueue* job_queue, size_t* res) {

  __shared__ Graph graph;
  __shared__ Pattern pat;
  __shared__ CallStack stk[NWARPS_PER_BLOCK];
  __shared__ size_t count[NWARPS_PER_BLOCK];

  int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
  int global_wid = global_tid / WARP_SIZE;
  int local_wid = threadIdx.x / WARP_SIZE;

  if (threadIdx.x == 0) {
    graph = *dev_graph;
    pat = *dev_pattern;
  }
  __syncthreads();

  if (threadIdx.x % WARP_SIZE == 0) {
    stk[local_wid] = dev_callstack[global_wid];
  }
  __syncwarp();

  auto start = clock64();
  while (true) {
    match(&graph, &pat, &stk[local_wid], job_queue, &count[local_wid]);
    break;
    // TODO: load balance
  }
  auto stop = clock64();

  if (threadIdx.x % WARP_SIZE == 0) {
    res[global_wid] = count[local_wid];
    //printf("%ld\n", stop - start);
  }
}
}