#include "hip/hip_runtime.h"
#include "gpu_match.cuh"
#include <hip/hip_runtime.h>

#define UNROLL_SIZE(l) (l > 0 ? UNROLL: 1) 

namespace STMatch {
  __device__ int gpuIdx;

  struct StealingArgs {
    int* idle_warps;
    int* idle_warps_count;
    int* global_mutex;
    int* local_mutex;
    CallStack* global_callstack;
  };

  __forceinline__ __device__ void lock(int* mutex) {
    while (atomicCAS((int*)mutex, 0, 1) != 0) {
    }
  }
  __forceinline__ __device__ void unlock(int* mutex) {
    atomicExch((int*)mutex, 0);
  }

  __device__ bool trans_layer(CallStack& _target_stk, CallStack& _cur_stk, Pattern* _pat, int _k, int ratio = 2) {
    if (_target_stk.level <= _k)
      return false;

    int num_left_task = _target_stk.slot_size[_pat->rowptr[_k]][_target_stk.uiter[_k]] -
      (_target_stk.iter[_k] + _target_stk.uiter[_k + 1] + 1);
    if (num_left_task <= 0)
      return false;

    int stealed_start_idx_in_target = _target_stk.iter[_k] + _target_stk.uiter[_k + 1] + 1 + num_left_task / ratio;

    _cur_stk.slot_storage[_pat->rowptr[0]][_target_stk.uiter[0]][_target_stk.iter[0] + _target_stk.uiter[1]] = _target_stk.slot_storage[_pat->rowptr[0]][_target_stk.uiter[0]][_target_stk.iter[0] + _target_stk.uiter[1]];
    _cur_stk.slot_storage[_pat->rowptr[0]][_target_stk.uiter[0]][_target_stk.iter[0] + _target_stk.uiter[1] + JOB_CHUNK_SIZE] = _target_stk.slot_storage[_pat->rowptr[0]][_target_stk.uiter[0]][_target_stk.iter[0] + _target_stk.uiter[1] + JOB_CHUNK_SIZE];

    for (int i = 1; i < _k; i++) {
      _cur_stk.slot_storage[_pat->rowptr[i]][_target_stk.uiter[i]][_target_stk.iter[i] + _target_stk.uiter[i + 1]] = _target_stk.slot_storage[_pat->rowptr[i]][_target_stk.uiter[i]][_target_stk.iter[i] + _target_stk.uiter[i + 1]];
    }

    for (int r = _pat->rowptr[_k]; r < _pat->rowptr[_k + 1]; r++) {
      for (int u = 0; u < UNROLL_SIZE(_k); u++) {
        int loop_end = _k == 0 ? JOB_CHUNK_SIZE * 2 : _target_stk.slot_size[r][u];
        for (int t = 0; t < loop_end; t++) {
          _cur_stk.slot_storage[r][u][t] = _target_stk.slot_storage[r][u][t];
        }
      }
    }

    for (int l = 0; l < _k; l++) {
      _cur_stk.iter[l] = _target_stk.iter[l];
      _cur_stk.uiter[l] = _target_stk.uiter[l];
      for (int s = _pat->rowptr[l]; s < _pat->rowptr[l + 1]; s++) {
        if (s > _pat->rowptr[l]) {
          for (int u = 0; u < UNROLL; u++) {
            _cur_stk.slot_size[s][u] = _target_stk.slot_size[s][u];
          }
        }
        else {
          for (int u = 0; u < UNROLL_SIZE(l); u++) {
            if (u == _cur_stk.uiter[l])
              _cur_stk.slot_size[_pat->rowptr[l]][u] = _target_stk.iter[l] + 1;
            else
              _cur_stk.slot_size[_pat->rowptr[l]][u] = 0;
          }
        }
      }
    }

    // copy
    for (int i = stealed_start_idx_in_target - _target_stk.iter[_k]; i < UNROLL_SIZE(_k + 1); i++) {
      _target_stk.slot_size[_pat->rowptr[_k + 1]][i] = 0;
    }

    for (int s = _pat->rowptr[_k]; s < _pat->rowptr[_k + 1]; s++) {
      if (s == _pat->rowptr[_k]) {
        for (int u = 0; u < UNROLL_SIZE(_k); u++) {
          if (u == _target_stk.uiter[_k])
            _cur_stk.slot_size[s][u] = _target_stk.slot_size[s][u];
          else
            _cur_stk.slot_size[s][u] = 0;
        }
      }
      else {
        for (int u = 0; u < UNROLL_SIZE(_k); u++) {
          _cur_stk.slot_size[s][u] = _target_stk.slot_size[s][u];
        }
      }
    }

    _cur_stk.uiter[_k] = _target_stk.uiter[_k];
    _cur_stk.iter[_k] = stealed_start_idx_in_target;
    _target_stk.slot_size[_pat->rowptr[_k]][_target_stk.uiter[_k]] = stealed_start_idx_in_target;
    // copy
    for (int l = _k + 1; l < _pat->nnodes - 1; l++) {
      _cur_stk.iter[l] = 0;
      _cur_stk.uiter[l] = 0;
      for (int s = _pat->rowptr[l]; s < _pat->rowptr[l + 1]; s++) {
        for (int u = 0; u < UNROLL_SIZE(l); u++) {
          _cur_stk.slot_size[s][u] = 0;
        }
      }
    }
    _cur_stk.iter[_pat->nnodes - 1] = 0;
    _cur_stk.uiter[_pat->nnodes - 1] = 0;
    for (int u = 0; u < UNROLL_SIZE(_pat->nnodes - 1); u++) {
      _cur_stk.slot_size[_pat->rowptr[_pat->nnodes - 1]][u] = 0;
    }
    _cur_stk.level = _k + 1;
    return true;
  }

  __device__ bool trans_skt(CallStack* _all_stk, CallStack* _cur_stk, Pattern* pat, StealingArgs* _stealing_args) {

    int max_left_task = 0;
    int stk_idx = -1;
    int at_level = -1;

    for (int level = 0; level < STOP_LEVEL; level++) {
      for (int i = 0; i < NWARPS_PER_BLOCK; i++) {

        if (i == threadIdx.x / WARP_SIZE)
          continue;
        lock(&(_stealing_args->local_mutex[i]));

        int left_task = _all_stk[i].slot_size[pat->rowptr[level]][_all_stk[i].uiter[level]] -
          (_all_stk[i].iter[level] + _all_stk[i].uiter[level + 1] + 1);
        if (left_task > max_left_task) {
          max_left_task = left_task;
          stk_idx = i;
          at_level = level;
        }
        unlock(&(_stealing_args->local_mutex[i]));
      }
      if (stk_idx != -1)
        break;
    }

    if (stk_idx != -1) {
      bool res;
      lock(&(_stealing_args->local_mutex[threadIdx.x / WARP_SIZE]));
      lock(&(_stealing_args->local_mutex[stk_idx]));
      res = trans_layer(_all_stk[stk_idx], *_cur_stk, pat, at_level);

      unlock(&(_stealing_args->local_mutex[threadIdx.x / WARP_SIZE]));
      unlock(&(_stealing_args->local_mutex[stk_idx]));
      return res;
    }
    return false;
  }


  __forceinline__ __device__ graph_node_t path(CallStack* stk, Pattern* pat, int level, int k) {
    if (level > 0)
      return stk->slot_storage[pat->rowptr[level]][stk->uiter[level]][stk->iter[level] + k];
    else {
      return stk->slot_storage[0][stk->uiter[0]][stk->iter[0] + k + (level + 1) * JOB_CHUNK_SIZE];
    }
  }

  typedef struct {
    graph_node_t* set1[UNROLL], * set2[UNROLL], * res[UNROLL];
    graph_node_t set1_size[UNROLL], set2_size[UNROLL], * res_size[UNROLL];
    graph_node_t ub[UNROLL];
    bitarray32 label;
    Graph* g;
    int num_sets;
    bool cached;
    int level;
    Pattern* pat;
  } Arg_t;

  template<typename DATA_T, typename SIZE_T>
  __forceinline__ __device__
    bool bsearch_exist(DATA_T* set2, SIZE_T set2_size, DATA_T target) {
    if (set2_size <= 0) return false;
    int mid;
    int low = 0;
    int high = set2_size - 1;
    while (low <= high) {
      mid = (low + high) / 2;
      if (target == set2[mid]) {
        return true;
      }
      else if (target > set2[mid]) {
        low = mid + 1;
      }
      else {
        high = mid - 1;
      }
    }
    return false;
  }

  template<typename DATA_T, typename SIZE_T>
  __forceinline__ __device__
    SIZE_T upper_bound(DATA_T* set2, SIZE_T set2_size, DATA_T target) {
    int i, step;
    int low = 0;
    while (set2_size > 0) {
      i = low;
      step = set2_size / 2;
      i += step;
      if (target > set2[i]) {
        low = ++i; set2_size -= step + 1;
      }
      else {
        set2_size = step;
      }
    }
    return low;
  }

  __forceinline__ __device__
    void prefix_sum(int* _input, int input_size) {

    int thid = threadIdx.x % WARP_SIZE;
    int offset = 1;
    int last_element = _input[input_size - 1];
    // build sum in place up the tree
    for (int d = (WARP_SIZE >> 1); d > 0; d >>= 1) {
      if (thid < d) {
        int ai = offset * (2 * thid + 1) - 1;
        int bi = offset * (2 * thid + 2) - 1;
        _input[bi] += _input[ai];
      }
      offset <<= 1;
    }
    if (thid == 0) { _input[WARP_SIZE - 1] = 0; } // clear the last element
     // traverse down tree & build scan
    for (int d = 1; d < WARP_SIZE; d <<= 1) {
      offset >>= 1;
      if (thid < d) {
        int ai = offset * (2 * thid + 1) - 1;
        int bi = offset * (2 * thid + 2) - 1;
        int t = _input[ai];
        _input[ai] = _input[bi];
        _input[bi] += t;
      }
    }
    __syncwarp();

    if (thid >= input_size - 1)
      _input[thid + 1] = _input[input_size - 1] + last_element;
  }


  template<bool DIFF>
  __device__ void compute_set(Arg_t* arg) {
    __shared__ graph_node_t size_psum[NWARPS_PER_BLOCK][WARP_SIZE + 1];
    __shared__ int end_pos[NWARPS_PER_BLOCK][UNROLL];

    int wid = threadIdx.x / WARP_SIZE;
    int tid = threadIdx.x % WARP_SIZE;

    if (tid < arg->num_sets) {
      arg->set1_size[tid] = upper_bound(arg->set1[tid], arg->set1_size[tid], arg->ub[tid]);
      size_psum[wid][tid] = arg->set1_size[tid];
      end_pos[wid][tid] = 0;
    }
    else {
      size_psum[wid][tid] = 0;
    }
    __syncwarp();

    prefix_sum(&size_psum[wid][0], arg->num_sets);
    __syncwarp();


    bool still_loop = true;
    int slot_idx = 0;
    int offset = 0;
    int predicate;

    //if(tid==0 && size_psum[wid][WARP_SIZE]>0){
    //    profPtr->busyThreadUsed[blockIdx.x][wid] += size_psum[wid][WARP_SIZE];
    //    profPtr->totalThreadUsed[blockIdx.x][wid] += (((size_psum[wid][WARP_SIZE] - 1) / WARP_SIZE + 1) * WARP_SIZE);
    //}    

    for (int idx = tid; (idx < ((size_psum[wid][WARP_SIZE] > 0) ? (((size_psum[wid][WARP_SIZE] - 1) / WARP_SIZE + 1) * WARP_SIZE) : 0) && still_loop); idx += WARP_SIZE) {
      predicate = 0;

      if (idx < size_psum[wid][WARP_SIZE]) {

        while (idx >= size_psum[wid][slot_idx + 1]) {
          slot_idx++;
        }
        offset = idx - size_psum[wid][slot_idx];

        bitarray32 lb = arg->g->vertex_label[arg->set1[slot_idx][offset]];

        predicate = ((lb & arg->label) == lb) && (DIFF ^ bsearch_exist(arg->set2[slot_idx], arg->set2_size[slot_idx], arg->set1[slot_idx][offset]));

      }
      else {
        slot_idx = arg->num_sets;
        still_loop = false;
      }

      still_loop = __shfl_sync(0xFFFFFFFF, still_loop, 31);
      predicate = __ballot_sync(0xFFFFFFFF, predicate);

      bool cond = (arg->level < arg->pat->nnodes - 2 && predicate & (1 << tid));
      graph_node_t res_tmp;
      if (cond) {
        res_tmp = arg->set1[slot_idx][offset];
      }

      int prev_idx = ((idx / WARP_SIZE == size_psum[wid][slot_idx] / WARP_SIZE) ? size_psum[wid][slot_idx] % WARP_SIZE : 0);

      if (cond) {
        arg->res[slot_idx][end_pos[wid][slot_idx] + __popc(predicate & ((1 << tid) - (1 << prev_idx)))] = res_tmp;
      }

      if (slot_idx < __shfl_down_sync(0xFFFFFFFF, slot_idx, 1)) {
        end_pos[wid][slot_idx] += __popc(predicate & ((1 << (tid + 1)) - (1 << prev_idx)));
      }
      else if (tid == WARP_SIZE - 1 && slot_idx < arg->num_sets) {
        end_pos[wid][slot_idx] += __popc(predicate & (0xFFFFFFFF - (1 << prev_idx) + 1));
      }
    }
    __syncwarp();
    if (tid < arg->num_sets) {
      *(arg->res_size[tid]) = end_pos[wid][tid];
    }
    __syncwarp();
  }

  __forceinline__ __device__ void get_job(JobQueue* q, graph_node_t& cur_pos, graph_node_t& njobs) {
    lock(&(q->mutex));
    cur_pos = q->cur;
    q->cur += JOB_CHUNK_SIZE;
    if (q->cur > q->length) q->cur = q->length;
    njobs = q->cur - cur_pos;
    unlock(&(q->mutex));
  }

  __device__ void extend(Graph* g, Pattern* pat, CallStack* stk, JobQueue* q, pattern_node_t level) {

    __shared__ Arg_t arg[NWARPS_PER_BLOCK];
    int wid = threadIdx.x / WARP_SIZE;

    if (level == 0) {
      graph_node_t cur_job, njobs;

      // TODO: change to warp
      for (int k = 0; k < UNROLL_SIZE(level); k++) {
        if (threadIdx.x % WARP_SIZE == 0) {
          get_job(q, cur_job, njobs);

          for (size_t i = 0; i < njobs; i++) {
            for (int j = 0; j < 2; j++) {
              stk->slot_storage[0][k][i + JOB_CHUNK_SIZE * j] = (q->q[cur_job + i].nodes)[j];
            }
          }
          stk->slot_size[0][k] = njobs;
        }
        __syncwarp();
      }
    }
    else {

      arg[wid].g = g;
      arg[wid].num_sets = UNROLL_SIZE(level);

      int remaining = stk->slot_size[pat->rowptr[level - 1]][stk->uiter[level - 1]] - stk->iter[level - 1];
      if (remaining >= 0 && UNROLL_SIZE(level) > remaining) {
        arg[wid].num_sets = remaining;
      }

      for (int i = pat->rowptr[level]; i < pat->rowptr[level + 1]; i++) {

        // compute ub based on pattern->partial
        if (!LABELED) {
          graph_node_t ub = ((i == pat->rowptr[level]) ? INT_MAX : -1);
          if (pat->partial[i] != 0) {

            // compute ub with nodes after start_level until previous level
            for (pattern_node_t k = 1; k < level - 1; k++) {
              if ((pat->partial[i] & (1 << (k + 1))) && ((i == pat->rowptr[level]) ^ (ub < path(stk, pat, k, stk->uiter[k + 1])))) ub = path(stk, pat, k, stk->uiter[k + 1]);
            }
            // compute ub with nodes in the previous level
            for (pattern_node_t k = 0; k < arg[wid].num_sets; k++) {
              arg[wid].ub[k] = ub;
              int prev_level = (level > 1 ? 2 : 1);
              int prev_iter = (level > 1 ? stk->uiter[1] : k);
              // compute ub with the first few nodes before start_level
              for (pattern_node_t j = 0; j < prev_level; j++) {
                if ((pat->partial[i] & (1 << j)) && ((i == pat->rowptr[level]) ^ (arg[wid].ub[k] < path(stk, pat, j - 1, prev_iter)))) arg[wid].ub[k] = path(stk, pat, j - 1, prev_iter);
              }

              if ((pat->partial[i] & (1 << level)) && ((i == pat->rowptr[level]) ^ (arg[wid].ub[k] < path(stk, pat, level - 1, k)))) arg[wid].ub[k] = path(stk, pat, level - 1, k);
              if (arg[wid].ub[k] == -1) arg[wid].ub[k] = INT_MAX;
            }
          }
          else {
            for (pattern_node_t k = 0; k < arg[wid].num_sets; k++) {
              arg[wid].ub[k] = INT_MAX;
            }
          }
        }
        else {
          for (pattern_node_t k = 0; k < arg[wid].num_sets; k++) {
            arg[wid].ub[k] = INT_MAX;
          }
        }

        arg[wid].label = pat->slot_labels[i];

        if (pat->set_ops[i] & 0x20) {

          for (graph_node_t k = 0; k < arg[wid].num_sets; k++) {

            arg[wid].set2[k] = NULL;
            arg[wid].set2_size[k] = 0;

            if (!EDGE_INDUCED) {
              graph_node_t t = path(stk, pat, level - 2, ((level > 1) ? stk->uiter[level - 1] : k));
              arg[wid].set2[k] = &g->colidx[g->rowptr[t]];
              arg[wid].set2_size[k] = (graph_node_t)(g->rowptr[t + 1] - g->rowptr[t]);
            }
            graph_node_t t = path(stk, pat, level - 1, k);
            arg[wid].set1[k] = &g->colidx[g->rowptr[t]];
            arg[wid].res[k] = &(stk->slot_storage[i][k][0]);
            arg[wid].set1_size[k] = (graph_node_t)(g->rowptr[t + 1] - g->rowptr[t]);
            arg[wid].res_size[k] = &(stk->slot_size[i][k]);
          }
          // arg[wid].cached = (level > 1);
          arg[wid].level = level;
          arg[wid].pat = pat;
          compute_set<true>(&arg[wid]);

          if (!EDGE_INDUCED) {
            for (pattern_node_t j = level - 3; j >= -1; j--) {
              graph_node_t t = path(stk, pat, j, stk->uiter[(j > 0 ? j + 1 : 1)]);

              for (graph_node_t k = 0; k < arg[wid].num_sets; k++) {
                arg[wid].set1[k] = &(stk->slot_storage[i][k][0]);
                arg[wid].set2[k] = &g->colidx[g->rowptr[t]];
                arg[wid].res[k] = &(stk->slot_storage[i][k][0]);
                arg[wid].set1_size[k] = stk->slot_size[i][k];
                arg[wid].set2_size[k] = (graph_node_t)(g->rowptr[t + 1] - g->rowptr[t]);
                arg[wid].res_size[k] = &(stk->slot_size[i][k]);
              }
              //arg[wid].cached = true;
              arg[wid].level = level;
              arg[wid].pat = pat;
              compute_set<true>(&arg[wid]);
            }
          }
          for (graph_node_t k = arg[wid].num_sets; k < UNROLL_SIZE(level); k++) stk->slot_size[i][k] = 0;
        }
        else {

          pattern_node_t slot_idx = (pat->set_ops[i] & 0x1F);

          if (pat->set_ops[i] & 0x40) {
            for (graph_node_t k = 0; k < arg[wid].num_sets; k++) {
              graph_node_t t = path(stk, pat, level - 1, k);
              graph_node_t* neighbor = &g->colidx[g->rowptr[t]];
              graph_node_t neighbor_size = (graph_node_t)(g->rowptr[t + 1] - g->rowptr[t]);

              if (level > 1) {
                arg[wid].set2[k] = &(stk->slot_storage[slot_idx][stk->uiter[level - 1]][0]);
                arg[wid].set2_size[k] = stk->slot_size[slot_idx][stk->uiter[level - 1]];
              }
              else {
                graph_node_t t = path(stk, pat, -1, k);
                arg[wid].set2[k] = &g->colidx[g->rowptr[t]];
                arg[wid].set2_size[k] = (graph_node_t)(g->rowptr[t + 1] - g->rowptr[t]);
              }

              arg[wid].set1[k] = neighbor;
              arg[wid].set1_size[k] = neighbor_size;
              arg[wid].res[k] = &(stk->slot_storage[i][k][0]);
              arg[wid].res_size[k] = &(stk->slot_size[i][k]);
            }
            //arg[wid].cached = (level > 1);
            arg[wid].level = level;
            arg[wid].pat = pat;
            compute_set<false>(&arg[wid]);
            for (graph_node_t k = arg[wid].num_sets; k < UNROLL_SIZE(level); k++) stk->slot_size[i][k] = 0;

          }
          else {

            for (graph_node_t k = 0; k < arg[wid].num_sets; k++) {
              graph_node_t* neighbor = NULL;
              graph_node_t neighbor_size = 0;
              if (!EDGE_INDUCED) {
                graph_node_t t = path(stk, pat, level - 1, k);
                neighbor = &g->colidx[g->rowptr[t]];
                neighbor_size = (graph_node_t)(g->rowptr[t + 1] - g->rowptr[t]);
              }

              if (level > 1) {
                arg[wid].set1[k] = &(stk->slot_storage[slot_idx][stk->uiter[level - 1]][0]);
                arg[wid].set1_size[k] = stk->slot_size[slot_idx][stk->uiter[level - 1]];
              }
              else {
                graph_node_t t = path(stk, pat, -1, k);
                arg[wid].set1[k] = &g->colidx[g->rowptr[t]];
                arg[wid].set1_size[k] = (graph_node_t)(g->rowptr[t + 1] - g->rowptr[t]);
              }

              arg[wid].set2[k] = neighbor;
              arg[wid].set2_size[k] = neighbor_size;
              arg[wid].res[k] = &(stk->slot_storage[i][k][0]);
              arg[wid].res_size[k] = &(stk->slot_size[i][k]);
            }
            //arg[wid].cached = false;
            arg[wid].level = level;
            arg[wid].pat = pat;
            compute_set<true>(&arg[wid]);
            for (graph_node_t k = arg[wid].num_sets; k < UNROLL_SIZE(level); k++) stk->slot_size[i][k] = 0;

          }
        }
      }
    }
    stk->iter[level] = 0;
    stk->uiter[level] = 0;
  }

  __forceinline__ __device__ void respond_across_block(int level, CallStack* stk, Pattern* pat, StealingArgs* _stealing_args) {
    if (level > 0 && level <= DETECT_LEVEL) {
      if (threadIdx.x % WARP_SIZE == 0) {
        int at_level = -1;
        int left_task = 0;
        for (int l = 0; l < level; l++) {
          left_task = stk->slot_size[pat->rowptr[l]][stk->uiter[l]] - stk->iter[l] - stk->uiter[l + 1] - 1;
          if (left_task > 0) {
            at_level = l;
            break;
          }
        }
        if (at_level != -1) {
          for (int b = 0; b < GRID_DIM; b++) {
            if (b == blockIdx.x) continue;
            if (atomicCAS(&(_stealing_args->global_mutex[b]), 0, 1) == 0) {
              if (atomicAdd(&_stealing_args->idle_warps[b], 0) == 0xFFFFFFFF) {
                __threadfence();

                trans_layer(*stk, _stealing_args->global_callstack[b * NWARPS_PER_BLOCK], pat, at_level, INT_MAX);
                __threadfence();

                atomicSub(_stealing_args->idle_warps_count, NWARPS_PER_BLOCK);
                atomicExch(&_stealing_args->idle_warps[b], 0);

                atomicExch(&(_stealing_args->global_mutex[b]), 0);
                break;
              }
              atomicExch(&(_stealing_args->global_mutex[b]), 0);
            }
          }
        }
      }
      __syncwarp();
    }
  }

  __device__ void match(Graph* g, Pattern* pat,
    CallStack* stk, JobQueue* q, size_t* count, StealingArgs* _stealing_args) {

    pattern_node_t& level = stk->level;

    while (true) {
      if (threadIdx.x % WARP_SIZE == 0) {
        lock(&(_stealing_args->local_mutex[threadIdx.x / WARP_SIZE]));
      }
      __syncwarp();

      if (level < pat->nnodes - 2) {

        if (STEAL_ACROSS_BLOCK) {
          respond_across_block(level, stk, pat, _stealing_args);
        }

        if (stk->uiter[level] == 0 && stk->slot_size[pat->rowptr[level]][0] == 0) {

          extend(g, pat, stk, q, level);
          if (level == 0 && stk->slot_size[0][0] == 0) {
            if (threadIdx.x % WARP_SIZE == 0)
              unlock(&(_stealing_args->local_mutex[threadIdx.x / WARP_SIZE]));
            __syncwarp();
            break;
          }
        }
        if (stk->uiter[level] < UNROLL_SIZE(level)) {
          if (stk->iter[level] < stk->slot_size[pat->rowptr[level]][stk->uiter[level]]) {
            if (threadIdx.x % WARP_SIZE == 0)
              level++;
            __syncwarp();
          }
          else {
            stk->slot_size[pat->rowptr[level]][stk->uiter[level]] = 0;
            stk->iter[level] = 0;
            if (threadIdx.x % WARP_SIZE == 0)
              stk->uiter[level]++;
            __syncwarp();
          }
        }
        else {
          stk->uiter[level] = 0;
          if (level > 0) {
            if (threadIdx.x % WARP_SIZE == 0)
              level--;
            if (threadIdx.x % WARP_SIZE == 0)
              stk->iter[level] += UNROLL_SIZE(level + 1);
            __syncwarp();
          }
        }
      }
      else if (level == pat->nnodes - 2) {

        extend(g, pat, stk, q, level);
        for (int j = 0; j < UNROLL_SIZE(level); j++) {
          if (threadIdx.x % WARP_SIZE == 0) {
            *count += stk->slot_size[pat->rowptr[level]][j];
          }
          __syncwarp();
          stk->slot_size[pat->rowptr[level]][j] = 0;
        }
        if (threadIdx.x % WARP_SIZE == 0)
          level--;
        if (threadIdx.x % WARP_SIZE == 0)
          stk->iter[level] += UNROLL_SIZE(level + 1);
        __syncwarp();
      }
      //__syncwarp();
      if (threadIdx.x % WARP_SIZE == 0)
        unlock(&(_stealing_args->local_mutex[threadIdx.x / WARP_SIZE]));
      __syncwarp();
    }
  }


/*
__device__ void getTransferedSlotSize(CallStack& stk, Pattern* _pat, bool isLocal){

    int _k = stk.level-1;
    
    size_t transferedSlotStorageSize = 0;
    for (int r = _pat->rowptr[_k]; r < _pat->rowptr[_k + 1]; r++) {
      for (int u = 0; u < UNROLL_SIZE(_k); u++) {
        int loop_end = _k == 0 ? JOB_CHUNK_SIZE * 2 : stk.slot_size[r][u];
        transferedSlotStorageSize+=loop_end*sizeof(graph_node_t);
      }
    }

    if(isLocal){
        profPtr->localMemStorage[blockIdx.x][threadIdx.x/WARP_SIZE] += transferedSlotStorageSize;
    }
    else{
       profPtr->globalMemStorage[blockIdx.x][threadIdx.x/WARP_SIZE] += transferedSlotStorageSize;
       profPtr->globalMemStk[blockIdx.x][threadIdx.x/WARP_SIZE] += sizeof(CallStack)*2;
    }
}

*/

  __global__ void _parallel_match(Graph* dev_graph, Pattern* dev_pattern,
    CallStack* dev_callstack, JobQueue* job_queue, size_t* res,
    int* idle_warps, int* idle_warps_count, int* global_mutex, int gpu_idx) 
  {
    gpuIdx = gpu_idx;
    __shared__ Graph graph;
    __shared__ Pattern pat;
    __shared__ CallStack stk[NWARPS_PER_BLOCK];
    __shared__ size_t count[NWARPS_PER_BLOCK];
    __shared__ bool stealed[NWARPS_PER_BLOCK];
    __shared__ int mutex_this_block[NWARPS_PER_BLOCK];

    __shared__ StealingArgs stealing_args;
    stealing_args.idle_warps = idle_warps;
    stealing_args.idle_warps_count = idle_warps_count;
    stealing_args.global_mutex = global_mutex;
    stealing_args.local_mutex = mutex_this_block;
    stealing_args.global_callstack = dev_callstack;

    int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
    int global_wid = global_tid / WARP_SIZE;
    int local_wid = threadIdx.x / WARP_SIZE;

    //if(global_tid==0) printf("gpu:%d, cur:%d, length:%d, end:%d\n", gpu_idx, job_queue->cur, job_queue->length, job_queue->cur+job_queue->length);
    if (threadIdx.x == 0) {
      graph = *dev_graph;
      pat = *dev_pattern;
    }
    __syncthreads();

    if (threadIdx.x % WARP_SIZE == 0) {

      stk[local_wid] = dev_callstack[global_wid];
    }
    __syncwarp();

    auto start = clock64();

    while (true) {
      match(&graph, &pat, &stk[local_wid], job_queue, &count[local_wid], &stealing_args);
      __syncwarp();

      stealed[local_wid] = false;

      if (STEAL_IN_BLOCK) {

        if (threadIdx.x % WARP_SIZE == 0) {
          stealed[local_wid] = trans_skt(stk, &stk[local_wid], &pat, &stealing_args);
          //getTransferedSlotSize(stk[local_wid], &pat, true);
        }
        __syncwarp();
      }

      if (STEAL_ACROSS_BLOCK) {
        
        if (!stealed[local_wid]) {

          __syncthreads();

          if (threadIdx.x % WARP_SIZE == 0) {

            atomicAdd(stealing_args.idle_warps_count, 1);

            lock(&(stealing_args.global_mutex[blockIdx.x]));

            atomicOr(&stealing_args.idle_warps[blockIdx.x], (1 << local_wid));

            unlock(&(stealing_args.global_mutex[blockIdx.x]));

            while ((atomicAdd(stealing_args.idle_warps_count, 0) < NWARPS_TOTAL) && (atomicAdd(&stealing_args.idle_warps[blockIdx.x], 0) & (1 << local_wid)));

           
            if (atomicAdd(stealing_args.idle_warps_count, 0) < NWARPS_TOTAL) {
              //printf("Global Stealed\n");
              __threadfence();
              if (local_wid == 0) {
                stk[local_wid] = (stealing_args.global_callstack[blockIdx.x * NWARPS_PER_BLOCK]);
               // getTransferedSlotSize(stk[local_wid], &pat, false);
              }
              stealed[local_wid] = true;
            }
            else {
              stealed[local_wid] = false;
            }
          }
          __syncthreads();
        }
      }

      if (!stealed[local_wid]) {
        break;
      }
    }

    auto stop = clock64();

    if (threadIdx.x % WARP_SIZE == 0) {
      res[global_wid] = count[local_wid];
      //res[global_wid] = stop - start;
       //printf("%d\t%ld\t%d\t%d\n", blockIdx.x, stop - start, stealed[local_wid], local_wid);
       //printf("%lld\n", stop-start);
    }
   // __syncwarp();
    //profPtr->clk[blockIdx.x][threadIdx.x] = stop - start;
  //if(global_tid==0){
  //  printf("%d gpu finished\n", gpu_idx);
 // }
  }
}
