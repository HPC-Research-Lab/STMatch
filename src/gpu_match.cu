#include "hip/hip_runtime.h"
#include "gpu_match.cuh"
#include <hip/hip_runtime.h>

// TODO: change this to gpu array so we can use different unroll for diffrent levels
//#define UNROLL_SIZE(l) UNROLL
#define UNROLL_SIZE(l) _unroll_size[l]

namespace libra {
  struct StealingArgs
  {
    unsigned int *idle_warps;
    int *idle_warps_count;
    int *global_mutex;
    int *local_mutex;
    bool *stk_valid;
    CallStack *global_callstack;
  };

  __device__ void lock(int* mutex) {
    while (atomicCAS((int*)mutex, 0, 1) != 0) {
    }
  }
  __device__ void unlock(int* mutex) {
    atomicExch((int*)mutex, 0);
  }

 
  __device__ bool trans_layer(CallStack &_target_stk, CallStack &_cur_stk, Pattern *_pat, int _k, int *_unroll_size)
  {
    if (_target_stk.level <= _k)
      return false;

    int num_left_task = _target_stk.slot_size[_pat->rowptr[_k]][_target_stk.uiter[_k]] -
                        (_target_stk.iter[_k] + _target_stk.uiter[_k + 1] + 1);
    if (num_left_task <= 0)
      return false;

    int stealed_start_idx_in_target = _target_stk.iter[_k] + _target_stk.uiter[_k + 1] + 1 + num_left_task / 2;

    // graph_node_t (*slot_storage)[UNROLL][GRAPH_DEGREE] = _cur_stk.slot_storage;
    //_cur_stk = _target_stk;
    //_cur_stk.slot_storage = slot_storage;
    // printf("_cur:%d, _stk_idx:%d, _k:%d,  _target_stk.level:%d\n",
    //         threadIdx.x / WARP_SIZE,
    //         _stk_idx,
    //         _k,
    //         _target_stk.level);

    // printf("slot_size:%d, uiter:%d,  iter:%d, uiter_k+1:%d, slot_size_k+1:%d, num_left_task:%d, stealed_start_idx_in_target:%d\n",
    //       _target_stk.slot_size[_pat->rowptr[_k]][_target_stk.uiter[_k]],
    //        _target_stk.uiter[_k],
    //        _target_stk.iter[_k],
    //        _target_stk.uiter[_k+1],
    //        _target_stk.slot_size[_pat->rowptr[_k+1]][_target_stk.uiter[_k+1]],
    //        num_left_task,
    //        stealed_start_idx_in_target
    //);
    for (int i = 0; i <= _k; i++)
    {
      memcpy((_cur_stk.slot_storage[_pat->rowptr[i]]), (_target_stk.slot_storage[_pat->rowptr[i]]), (_pat->rowptr[i + 1] - _pat->rowptr[i]) * GRAPH_DEGREE * UNROLL_SIZE(i) * sizeof(graph_node_t));
    }

    // Copy

    for (int l = 0; l < _k; l++)
    {
      _cur_stk.iter[l] = _target_stk.iter[l];
      _cur_stk.uiter[l] = _target_stk.uiter[l];
      for (int s = _pat->rowptr[l]; s < _pat->rowptr[l + 1]; s++)
      {
        for (int u = 0; u < UNROLL_SIZE(l); u++)
        {
          _cur_stk.slot_size[s][u] = _target_stk.slot_size[s][u];
        }
      }
    }

    // copy
    for (int i = stealed_start_idx_in_target - _target_stk.iter[_k]; i < UNROLL_SIZE(_k + 1); i++)
    {
      _target_stk.slot_size[_pat->rowptr[_k + 1]][i] = 0;
    }

    for (int s = _pat->rowptr[_k]; s < _pat->rowptr[_k + 1]; s++)
    {
      if (s == _pat->rowptr[_k])
      {
        for (int u = 0; u < UNROLL_SIZE(_k); u++)
        {
          if (u == _target_stk.uiter[_k])
            _cur_stk.slot_size[s][u] = _target_stk.slot_size[s][u];
          else
            _cur_stk.slot_size[s][u] = 0;
        }
      }
      else
      {
        for (int u = 0; u < UNROLL_SIZE(_k); u++)
        {
          _cur_stk.slot_size[s][u] = _target_stk.slot_size[s][u];
        }
      }
    }

    _cur_stk.uiter[_k] = _target_stk.uiter[_k];
    _cur_stk.iter[_k] = stealed_start_idx_in_target;
    _target_stk.slot_size[_pat->rowptr[_k]][_target_stk.uiter[_k]] = stealed_start_idx_in_target;
    // copy
    for (int l = _k + 1; l < _pat->nnodes - 1; l++)
    {
      _cur_stk.iter[l] = 0;
      _cur_stk.uiter[l] = 0;
      for (int s = _pat->rowptr[l]; s < _pat->rowptr[l + 1]; s++)
      {
        for (int u = 0; u < UNROLL_SIZE(l); u++)
        {
          _cur_stk.slot_size[s][u] = 0;
        }
      }
    }
    _cur_stk.iter[_pat->nnodes - 1] = 0;
    _cur_stk.uiter[_pat->nnodes - 1] = 0;
    for (int u = 0; u < UNROLL_SIZE(_pat->nnodes - 1); u++)
    {
      _cur_stk.slot_size[_pat->rowptr[_pat->nnodes - 1]][u] = 0;
    }
    _cur_stk.level = _k + 1;
    _cur_stk.start_level = _target_stk.start_level;
    // return false;
    return true;
  }

  __device__ bool trans_skt(CallStack *_all_stk, CallStack *_cur_stk, Pattern *pat, int *_unroll_size, StealingArgs *_stealing_args)
  {
    // int local_mutex = 0;
    int max_left_task = 0;
    int stk_idx = -1;
    int at_level = -1;
    // int task_level = -1;

    // for(int level = 0; level< pat->nnodes - 2; level++){
    for (int level = 0; level < 1; level++)
    {
      // if(level==2) continue;
      for (int i = 0; i < NWARPS_PER_BLOCK; i++)
      {

        if (i == threadIdx.x / WARP_SIZE)
          continue;
        lock(&(_stealing_args->local_mutex[i]));
        // printf("slot_size 0:%d\n", _all_stk[i].slot_size[0][1]);
        // continue;

        if (_all_stk[i].level > level)
        {
          int left_task = _all_stk[i].slot_size[pat->rowptr[level]][_all_stk[i].uiter[level]] -
                          (_all_stk[i].iter[level] + _all_stk[i].uiter[level + 1] + 1);
          if (left_task > max_left_task)
          {
            max_left_task = left_task;
            stk_idx = i;
            at_level = level;
          }
        }
        else
        {
          unlock(&(_stealing_args->local_mutex[i]));
          continue;
        }
        unlock(&(_stealing_args->local_mutex[i]));
      }
      if (stk_idx != -1)
        break;
    }

    if (stk_idx != -1)
    {
      // printf("stk_idx:%d\n", stk_idx);
      int res = -1;
      lock(&(_stealing_args->local_mutex[threadIdx.x / WARP_SIZE]));
      lock(&(_stealing_args->local_mutex[stk_idx]));
      //__threadfence_block();
      res = trans_layer(_all_stk[stk_idx], *_cur_stk, pat, at_level, _unroll_size);

      unlock(&(_stealing_args->local_mutex[threadIdx.x / WARP_SIZE]));
      unlock(&(_stealing_args->local_mutex[stk_idx]));
      return res;
    }
    return false;
  }


  inline __device__ graph_node_t path(CallStack* stk, Pattern* pat, int level, int k) {
    if (level > 0)
      return stk->slot_storage[pat->rowptr[level]][stk->uiter[level]][stk->iter[level] + k];
    else {
      return stk->slot_storage[0][stk->uiter[0]][stk->iter[0] + k + (stk->start_level + level - 1) * JOB_CHUNK_SIZE];
    }
  }

  typedef struct {
    graph_node_t* set1[UNROLL], * set2[UNROLL], * res[UNROLL];
    graph_node_t set1_size[UNROLL], set2_size[UNROLL], * res_size[UNROLL];
    graph_node_t ub[UNROLL];
    bitarray32 label;
    Graph* g;
    int num_sets;
  } Arg_t;

  template<typename DATA_T, typename SIZE_T>
  inline __device__
    bool bsearch_exist(DATA_T* set2, SIZE_T set2_size, DATA_T target) {
    int mid;
    int low = 0;
    int high = set2_size - 1;
    while (low <= high) {
      mid = (low + high) / 2;
      if (target == set2[mid]) {
        return true;
      }
      else if (target > set2[mid]) {
        low = mid + 1;
      }
      else {
        high = mid - 1;
      }
    }
    return false;
  }

  template<typename DATA_T, typename SIZE_T>
  inline __device__
    SIZE_T upper_bound(DATA_T* set2, SIZE_T set2_size, DATA_T target) {
    int i, step;
    int low = 0;
    while (set2_size > 0) {
      i = low;
      step = set2_size / 2;
      i += step;
      if (target > set2[i]) {
        low = ++i; set2_size -= step + 1;
      }
      else {
        set2_size = step;
      }
    }
    return low;
  }

  inline __device__
    void prefix_sum(int* _input, int input_size) {

    int thid = threadIdx.x % WARP_SIZE;
    int offset = 1;
    int last_element = _input[input_size - 1];
    // build sum in place up the tree
    for (int d = (WARP_SIZE >> 1); d > 0; d >>= 1) {
      if (thid < d) {
        int ai = offset * (2 * thid + 1) - 1;
        int bi = offset * (2 * thid + 2) - 1;
        _input[bi] += _input[ai];
      }
      offset <<= 1;
    }
    if (thid == 0) { _input[WARP_SIZE - 1] = 0; } // clear the last element
     // traverse down tree & build scan
    for (int d = 1; d < WARP_SIZE; d <<= 1) {
      offset >>= 1;
      if (thid < d) {
        int ai = offset * (2 * thid + 1) - 1;
        int bi = offset * (2 * thid + 2) - 1;
        int t = _input[ai];
        _input[ai] = _input[bi];
        _input[bi] += t;
      }
    }
    __syncwarp();

    if (thid >= input_size - 1)
      _input[thid + 1] = _input[input_size - 1] + last_element;
  }


  template<bool DIFF>
  __device__ void compute_set(Arg_t* arg) {
    __shared__ int pos[NWARPS_PER_BLOCK][WARP_SIZE + 1];
    __shared__ graph_node_t size_psum[NWARPS_PER_BLOCK][WARP_SIZE + 1];
    __shared__ int end_pos[NWARPS_PER_BLOCK][UNROLL];

    int wid = threadIdx.x / WARP_SIZE;
    int tid = threadIdx.x % WARP_SIZE;


    if (tid < arg->num_sets) {
      arg->set1_size[tid] = upper_bound(arg->set1[tid], arg->set1_size[tid], arg->ub[tid]);
      size_psum[wid][tid] = arg->set1_size[tid];
      end_pos[wid][tid] = 0;
    }
    else {
      size_psum[wid][tid] = 0;
    }
    __syncwarp();

    prefix_sum(&size_psum[wid][0], arg->num_sets);
    __syncwarp();


    bool still_loop = true;
    int slot_idx = 0;
    int offset = 0;
    int predicate;

    int size1 = (size_psum[wid][WARP_SIZE] > 0) ? (((size_psum[wid][WARP_SIZE] - 1) / WARP_SIZE + 1) * WARP_SIZE) : 0;

    for (int idx = tid; (idx < size1 && still_loop); idx += WARP_SIZE) {
      predicate = 0;

      if (idx < size_psum[wid][WARP_SIZE]) {

        while (idx >= size_psum[wid][slot_idx + 1]) {
          slot_idx++;
        }
        offset = idx - size_psum[wid][slot_idx];

        bitarray32 lb = arg->g->vertex_label[arg->set1[slot_idx][offset]];
        if ((lb && arg->label == lb) && (DIFF ^ bsearch_exist(arg->set2[slot_idx], arg->set2_size[slot_idx], arg->set1[slot_idx][offset]))) {
          predicate = 1;
        }
      }
      else {
        slot_idx = arg->num_sets;
        still_loop = false;
      }

      still_loop = __shfl_sync(0xFFFFFFFF, still_loop, 31);
      predicate = __ballot_sync(0xFFFFFFFF, predicate);
      pos[wid][tid] = __popc(predicate & ((1 << tid) - 1));
      pos[wid][WARP_SIZE] = __popc(predicate);
      __syncwarp();


      graph_node_t res_tmp;
      if (pos[wid][tid + 1] > pos[wid][tid]) {
        res_tmp = arg->set1[slot_idx][offset];
      }
      __syncwarp();
      int prev_idx = ((idx / WARP_SIZE == size_psum[wid][slot_idx] / WARP_SIZE) ? size_psum[wid][slot_idx] % WARP_SIZE : 0);

      if (pos[wid][tid + 1] > pos[wid][tid]) {
        arg->res[slot_idx][end_pos[wid][slot_idx] + pos[wid][tid] - pos[wid][prev_idx]] = res_tmp;
      }

      if (slot_idx < __shfl_down_sync(0xFFFFFFFF, slot_idx, 1)) {
        end_pos[wid][slot_idx] += pos[wid][tid + 1] - pos[wid][prev_idx];
      }
      else if (tid == WARP_SIZE - 1 && slot_idx < arg->num_sets) {
        end_pos[wid][slot_idx] += pos[wid][WARP_SIZE] - pos[wid][prev_idx];
      }
    }
    __syncwarp();
    if (tid < arg->num_sets) {
      *(arg->res_size[tid]) = end_pos[wid][tid];
    }
    __syncwarp();
  }

  inline __device__ void get_job(JobQueue* q, graph_node_t& cur_pos, graph_node_t& njobs) {
    lock(&(q->mutex));
    cur_pos = q->cur;
    q->cur += JOB_CHUNK_SIZE;
    if (q->cur > q->length) q->cur = q->length;
    njobs = q->cur - cur_pos;
    unlock(&(q->mutex));
    // cur_pos = atomicAdd(&q->cur, JOB_CHUNK_SIZE);
    // if (cur_pos < q->length) {
    //   njobs = (cur_pos + JOB_CHUNK_SIZE > q->length) ?  q->length - cur_pos : JOB_CHUNK_SIZE;
    // }
    // else njobs = 0;
  }

  __device__ void extend(Graph* g, Pattern* pat, CallStack* stk, JobQueue* q, pattern_node_t level, int* _unroll_size) {

    __shared__ Arg_t arg[NWARPS_PER_BLOCK];
    int wid = threadIdx.x / WARP_SIZE;

    if (level == 0) {
      graph_node_t cur_job, njobs;

      // TODO: change to warp
      for (int k = 0; k < UNROLL_SIZE(level); k++) {
        if (threadIdx.x % WARP_SIZE == 0) {
          get_job(q, cur_job, njobs);

          for (size_t i = 0; i < njobs; i++) {
            for (int j = 0; j < q->start_level; j++) {
              stk->slot_storage[0][k][i + JOB_CHUNK_SIZE * j] = (q->q[cur_job + i].nodes)[j];
            }
          }
          stk->slot_size[0][k] = njobs;
          stk->start_level = q->start_level;
        }
        __syncwarp();
      }
    }
    else {

      arg[wid].g = g;
      arg[wid].num_sets = UNROLL_SIZE(level);

      int remaining = stk->slot_size[pat->rowptr[level - 1]][stk->uiter[level - 1]] - stk->iter[level - 1];
      if (remaining >= 0 && UNROLL_SIZE(level) > remaining) {
        arg[wid].num_sets = remaining;
      }

      for (int i = pat->rowptr[level]; i < pat->rowptr[level + 1]; i++) {

        // compute ub based on pattern->partial
        graph_node_t ub = ((i == pat->rowptr[level]) ? INT_MAX : -1);
        if (pat->partial[i] != 0) {

          // compute ub with nodes after start_level until previous level
          for (pattern_node_t k = 1; k < level - 1; k++) {
            if ((pat->partial[i] & (1 << (k + stk->start_level - 1))) && ((i == pat->rowptr[level]) ^ (ub < path(stk, pat, k, stk->uiter[k + 1])))) ub = path(stk, pat, k, stk->uiter[k + 1]);
          }
          // compute ub with nodes in the previous level
          for (pattern_node_t k = 0; k < arg[wid].num_sets; k++) {
            arg[wid].ub[k] = ub;
            int prev_level = (level > 1 ? stk->start_level : stk->start_level - 1);
            int prev_iter = (level > 1 ? stk->uiter[1] : k);
            // compute ub with the first few nodes before start_level
            for (pattern_node_t j = 0; j < prev_level; j++) {
              if ((pat->partial[i] & (1 << j)) && ((i == pat->rowptr[level]) ^ (arg[wid].ub[k] < path(stk, pat, j + 1 - stk->start_level, prev_iter)))) arg[wid].ub[k] = path(stk, pat, j + 1 - stk->start_level, prev_iter);
            }

            if ((pat->partial[i] & (1 << (level + stk->start_level - 2))) && ((i == pat->rowptr[level]) ^ (arg[wid].ub[k] < path(stk, pat, level - 1, k)))) arg[wid].ub[k] = path(stk, pat, level - 1, k);
            if (arg[wid].ub[k] == -1) arg[wid].ub[k] = INT_MAX;
          }
        }
        else {
          for (pattern_node_t k = 0; k < arg[wid].num_sets; k++) {
            arg[wid].ub[k] = INT_MAX;
          }
        }

        arg[wid].label = pat->slot_labels[i];

        if (pat->set_ops[i] & 0x20) {

          for (graph_node_t k = 0; k < arg[wid].num_sets; k++) {

            arg[wid].set2[k] = NULL;
            arg[wid].set2_size[k] = 0;

            if (!EDGE_INDUCED) {
              graph_node_t t = path(stk, pat, level - 2, ((level > 1) ? stk->uiter[level - 1] : k));
              arg[wid].set2[k] = &g->colidx[g->rowptr[t]];
              arg[wid].set2_size[k] = (graph_node_t)(g->rowptr[t + 1] - g->rowptr[t]);
            }
            graph_node_t t = path(stk, pat, level - 1, k);
            arg[wid].set1[k] = &g->colidx[g->rowptr[t]];
            arg[wid].res[k] = &(stk->slot_storage[i][k][0]);
            arg[wid].set1_size[k] = (graph_node_t)(g->rowptr[t + 1] - g->rowptr[t]);
            arg[wid].res_size[k] = &(stk->slot_size[i][k]);
          }
          compute_set<true>(&arg[wid]);

          if (!EDGE_INDUCED) {
            for (pattern_node_t j = level - 3; j >= 1 - stk->start_level; j--) {
              graph_node_t t = path(stk, pat, j, stk->uiter[(j > 0 ? j + 1 : 1)]);

              for (graph_node_t k = 0; k < arg[wid].num_sets; k++) {
                arg[wid].set1[k] = &(stk->slot_storage[i][k][0]);
                arg[wid].set2[k] = &g->colidx[g->rowptr[t]];
                arg[wid].res[k] = &(stk->slot_storage[i][k][0]);
                arg[wid].set1_size[k] = stk->slot_size[i][k];
                arg[wid].set2_size[k] = (graph_node_t)(g->rowptr[t + 1] - g->rowptr[t]);
                arg[wid].res_size[k] = &(stk->slot_size[i][k]);
              }
              compute_set<true>(&arg[wid]);
            }
          }
          for (graph_node_t k = arg[wid].num_sets; k < UNROLL_SIZE(level); k++) stk->slot_size[i][k] = 0;
        }
        else {

          pattern_node_t slot_idx = (pat->set_ops[i] & 0x1F);

          if (pat->set_ops[i] & 0x40) {
            for (graph_node_t k = 0; k < arg[wid].num_sets; k++) {
              graph_node_t t = path(stk, pat, level - 1, k);
              graph_node_t* neighbor = &g->colidx[g->rowptr[t]];
              graph_node_t neighbor_size = (graph_node_t)(g->rowptr[t + 1] - g->rowptr[t]);

              if (level > 1) {
                arg[wid].set1[k] = &(stk->slot_storage[slot_idx][stk->uiter[level - 1]][0]);
                arg[wid].set1_size[k] = stk->slot_size[slot_idx][stk->uiter[level - 1]];
              }
              else {
                graph_node_t t = path(stk, pat, -1, k);
                arg[wid].set1[k] = &g->colidx[g->rowptr[t]];
                arg[wid].set1_size[k] = (graph_node_t)(g->rowptr[t + 1] - g->rowptr[t]);
              }

              arg[wid].set2[k] = neighbor;
              arg[wid].set2_size[k] = neighbor_size;
              arg[wid].res[k] = &(stk->slot_storage[i][k][0]);
              arg[wid].res_size[k] = &(stk->slot_size[i][k]);
            }
            compute_set<false>(&arg[wid]);
            for (graph_node_t k = arg[wid].num_sets; k < UNROLL_SIZE(level); k++) stk->slot_size[i][k] = 0;

          }
          else {

            for (graph_node_t k = 0; k < arg[wid].num_sets; k++) {
              graph_node_t* neighbor = NULL;
              graph_node_t neighbor_size = 0;
              if (!EDGE_INDUCED) {
                graph_node_t t = path(stk, pat, level - 1, k);
                neighbor = &g->colidx[g->rowptr[t]];
                neighbor_size = (graph_node_t)(g->rowptr[t + 1] - g->rowptr[t]);
              }

              if (level > 1) {
                arg[wid].set1[k] = &(stk->slot_storage[slot_idx][stk->uiter[level - 1]][0]);
                arg[wid].set1_size[k] = stk->slot_size[slot_idx][stk->uiter[level - 1]];
              }
              else {
                graph_node_t t = path(stk, pat, -1, k);
                arg[wid].set1[k] = &g->colidx[g->rowptr[t]];
                arg[wid].set1_size[k] = (graph_node_t)(g->rowptr[t + 1] - g->rowptr[t]);
              }

              arg[wid].set2[k] = neighbor;
              arg[wid].set2_size[k] = neighbor_size;
              arg[wid].res[k] = &(stk->slot_storage[i][k][0]);
              arg[wid].res_size[k] = &(stk->slot_size[i][k]);
            }
            compute_set<true>(&arg[wid]);
            for (graph_node_t k = arg[wid].num_sets; k < UNROLL_SIZE(level); k++) stk->slot_size[i][k] = 0;

          }
        }
      }
    }
    stk->iter[level] = 0;
    stk->uiter[level] = 0;
  }

  __device__ void match(Graph *g, Pattern *pat,
                        CallStack *stk, JobQueue *q, size_t *count, int *_unroll_size, StealingArgs *_stealing_args)
  {
    // if(threadIdx.x==0){
    //   printf("%d\n", UNROLL_SIZE(1));
    // }
    // pattern_node_t level = 0;
    // printf("stk->start_level:%d\n", stk->start_level);
    pattern_node_t &level = stk->level;

    __shared__ int target_block_idx[NWARPS_PER_BLOCK];

    while (true)
    {
      if (threadIdx.x % WARP_SIZE == 0)
      {
        lock(&(_stealing_args->local_mutex[threadIdx.x / WARP_SIZE]));
      }
      __syncwarp();

      if (level < pat->nnodes - stk->start_level)
      {

        if (level == 1)
        {
          if (threadIdx.x % WARP_SIZE == 0)
          {
            int left_task = stk->slot_size[pat->rowptr[level - 1]][stk->uiter[level - 1]] - stk->iter[level - 1] - stk->uiter[level] - 1;
            if (left_task > 0)
            {
              for (int b = blockIdx.x+1; b < GRID_DIM; b++)
              {
                  if (b == blockIdx.x || _stealing_args->global_mutex[b] == 1) continue;

                  lock(&(_stealing_args->global_mutex[b]));
                  if (_stealing_args->idle_warps[b] ==0xFFFFFFFF && !_stealing_args->stk_valid[b] )
                  {
                      printf("Find Idle Block:%d\n", b);
                      
                      trans_layer(*stk, _stealing_args->global_callstack[b * NWARPS_PER_BLOCK], pat, 0, _unroll_size);
                      _stealing_args->stk_valid[b] = true;
                      
                      atomicSub(_stealing_args->idle_warps_count, NWARPS_PER_BLOCK);
                      _stealing_args->idle_warps[b] = 0;
                      
                      unlock(&(_stealing_args->global_mutex[b]));
                      __threadfence();
                      break;
                  }
                  unlock(&(_stealing_args->global_mutex[b]));
                  __threadfence();
              }
            }
          }
          __syncwarp();
        }

        if (stk->uiter[level] == 0 && stk->slot_size[pat->rowptr[level]][0] == 0)
        {

          extend(g, pat, stk, q, level, _unroll_size);
          if (level == 0 && stk->slot_size[0][0] == 0)
          {
            if (threadIdx.x % WARP_SIZE == 0)
              unlock(&(_stealing_args->local_mutex[threadIdx.x / WARP_SIZE]));
            __syncwarp();
            break;
          }
        }
        if (stk->uiter[level] < UNROLL_SIZE(level))
        {
          if (stk->iter[level] < stk->slot_size[pat->rowptr[level]][stk->uiter[level]])
          {
            if (threadIdx.x % WARP_SIZE == 0)
              level++;
            __syncwarp();
          }
          else
          {
            stk->slot_size[pat->rowptr[level]][stk->uiter[level]] = 0;
            stk->iter[level] = 0;
            if (threadIdx.x % WARP_SIZE == 0)
              stk->uiter[level]++;
            __syncwarp();
          }
        }
        else
        {
          stk->uiter[level] = 0;
          if (level > 0)
          {
            if (threadIdx.x % WARP_SIZE == 0)
              level--;
            if (threadIdx.x % WARP_SIZE == 0)
              stk->iter[level] += UNROLL_SIZE(level + 1);
            __syncwarp();
          }
        }
      }
      else if (level == pat->nnodes - stk->start_level)
      {

        // TODO: we can save the storage of sets for the last level
        extend(g, pat, stk, q, level, _unroll_size);
        for (int j = 0; j < UNROLL_SIZE(level); j++)
        {
          if (threadIdx.x % WARP_SIZE == 0)
          {
            *count += stk->slot_size[pat->rowptr[level]][j];
          }
          __syncwarp();
          stk->slot_size[pat->rowptr[level]][j] = 0;
        }
        if (threadIdx.x % WARP_SIZE == 0)
          level--;
        if (threadIdx.x % WARP_SIZE == 0)
          stk->iter[level] += UNROLL_SIZE(level + 1);
        __syncwarp();
      }
      //__syncwarp();
      if (threadIdx.x % WARP_SIZE == 0)
        unlock(&(_stealing_args->local_mutex[threadIdx.x / WARP_SIZE]));
      __syncwarp();
    }
  }


 
  __global__ void _parallel_match(Graph *dev_graph, Pattern *dev_pattern,
                                  CallStack *dev_callstack, JobQueue *job_queue, size_t *res,
                                  unsigned int *idle_warps, int *idle_warps_count, int *global_mutex, bool *stk_valid)
  {
    __shared__ Graph graph;
    __shared__ Pattern pat;
    __shared__ CallStack stk[NWARPS_PER_BLOCK];
    __shared__ size_t count[NWARPS_PER_BLOCK];
    __shared__ bool stealed[NWARPS_PER_BLOCK];
    __shared__ int mutex_this_block[NWARPS_PER_BLOCK];
    __shared__ int unroll_size[PAT_SIZE];

    __shared__ StealingArgs stealing_args;
    stealing_args.idle_warps = idle_warps;
    stealing_args.idle_warps_count = idle_warps_count;
    stealing_args.global_mutex = global_mutex;
    stealing_args.local_mutex = mutex_this_block;
    stealing_args.stk_valid = stk_valid;
    stealing_args.global_callstack = dev_callstack;
    /*
      if(threadIdx.x % WARP_SIZE == 0){
        unroll_size[threadIdx.x % WARP_SIZE] = 1;
      }
      else if(threadIdx.x % WARP_SIZE == 1){
        unroll_size[threadIdx.x % WARP_SIZE] = 2;
      }
      else if(threadIdx.x % WARP_SIZE <PAT_SIZE){
         unroll_size[threadIdx.x % WARP_SIZE] = 4;
      }
      */

    int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
    int global_wid = global_tid / WARP_SIZE;
    int local_wid = threadIdx.x / WARP_SIZE;

    if (threadIdx.x == 0)
    {
      unroll_size[0] = 1;
      unroll_size[1] = 4;
      unroll_size[2] = 4;
      unroll_size[3] = 4;
      unroll_size[4] = 4;
      unroll_size[5] = 4;
      graph = *dev_graph;
      pat = *dev_pattern;
    }
    __syncthreads();

    if (threadIdx.x % WARP_SIZE == 0)
    {

      stk[local_wid] = dev_callstack[global_wid];
    }
    __syncwarp();

    auto start = clock64();

    while (true)
    {
      match(&graph, &pat, &stk[local_wid], job_queue, &count[local_wid], unroll_size, &stealing_args);
      __syncwarp();
      // break;
      // trans_success[local_wid]=false;
      if (threadIdx.x % WARP_SIZE == 0)
      {
        stealed[local_wid] = trans_skt(stk, &stk[local_wid], &pat, unroll_size, &stealing_args);
      }
      __syncwarp();
      if (!stealed[local_wid])
      {
         __syncthreads();
        if (threadIdx.x % WARP_SIZE == 0)
        {
          atomicAdd(stealing_args.idle_warps_count, 1);
          lock(&(stealing_args.global_mutex[blockIdx.x]));
          atomicOr(&(stealing_args.idle_warps[blockIdx.x]), (1 << local_wid));
          unlock(&(stealing_args.global_mutex[blockIdx.x]));
        }
        __syncthreads();
        if (threadIdx.x % WARP_SIZE == 0){
          // while(*(stealing_args.idle_warps_count) < NWARPS_TOTAL){
          //  __threadfence();
          //}
          while( (*stealing_args.idle_warps_count < NWARPS_TOTAL) && (stealing_args.idle_warps[blockIdx.x] & (1 << local_wid))){
              __threadfence();
          }
        }
        __syncthreads();

        if (threadIdx.x % WARP_SIZE == 0){
          if (*stealing_args.idle_warps_count < NWARPS_TOTAL)
          {
            lock(&(stealing_args.global_mutex[blockIdx.x]));
            
            if (stealing_args.stk_valid[blockIdx.x] == true)
            {
              __threadfence();
              stealing_args.stk_valid[blockIdx.x] = false;
              graph_node_t(*slot_storage_tmp)[UNROLL][GRAPH_DEGREE] = stk[local_wid].slot_storage;
              memcpy(stk[local_wid].slot_storage, stealing_args.global_callstack[blockIdx.x * NWARPS_PER_BLOCK].slot_storage, sizeof(graph_node_t) * MAX_SLOT_NUM * UNROLL * GRAPH_DEGREE);
               __threadfence();
              stk[local_wid] = (stealing_args.global_callstack[blockIdx.x * NWARPS_PER_BLOCK]);
              stk[local_wid].slot_storage = slot_storage_tmp;
               __threadfence();
            }
            
            stealed[local_wid] = true;
            unlock(&(stealing_args.global_mutex[blockIdx.x]));
             __threadfence();
          }
          else
          {
            stealed[local_wid] = false;
             __threadfence();
          }
        }
        __syncwarp();
        if (!stealed[local_wid])
        {
          break;
        }
        //__syncthreads();
      }
    }
    __syncthreads();

    auto stop = clock64();

    if (threadIdx.x % WARP_SIZE == 0)
    {
      res[global_wid] = count[local_wid];
      // printf("%d\t%ld\t%d\t%d\n", blockIdx.x, stop - start, stealed[local_wid], local_wid);
      // printf("%ld\n", stop - start);
    }

    // if(threadIdx.x % WARP_SIZE == 0)
    //   printf("%d\t%d\t%d\n", blockIdx.x, local_wid, mutex_this_block[local_wid]);
  }
}
