#include "hip/hip_runtime.h"
#include "gpu_match.cuh"
#include <hip/hip_runtime.h>

// TODO: change this to gpu array so we can use different unroll for diffrent levels
#define UNROLL_SIZE(l) 1

namespace libra {

  inline __device__ graph_node_t path(CallStack* stk, int level, int uiter) {
    if (level >= stk->start_level) {
      return stk->slot_storage[level][0][uiter][stk->iter[level]];
    }
    else {
      return stk->slot_storage[0][0][uiter][stk->iter[0] + JOB_CHUNK_SIZE * level];
    }
  }

  template<typename DATA_T, typename SIZE_T>
  __device__
    inline bool lower_bound_exist(DATA_T* set2, SIZE_T set2_size, DATA_T target) {
    int mid;
    int low = 0;
    int high = set2_size - 1;
    while (low <= high) {
      mid = (low + high) / 2;
      if (target == set2[mid]) {
        return true;
      }
      else if (target > set2[mid]) {
        low = mid + 1;
      }
      else {
        high = mid - 1;
      }
    }
    return false;
  }


  __device__ inline void prefix_sum(int* _input) {

    int thid = threadIdx.x % WARP_SIZE;
    int offset = 1;
    int last_element = _input[WARP_SIZE - 1];
    // build sum in place up the tree
    for (int d = (WARP_SIZE >> 1); d > 0; d >>= 1) {
      //__syncthreads();
      if (thid < d) {
        int ai = offset * (2 * thid + 1) - 1;
        int bi = offset * (2 * thid + 2) - 1;
        _input[bi] += _input[ai];
      }
      offset <<= 1;
    }
    if (thid == 0) { _input[WARP_SIZE - 1] = 0; } // clear the last element
     // traverse down tree & build scan
    for (int d = 1; d < WARP_SIZE; d <<= 1) {
      offset >>= 1;
      //__syncthreads();
      if (thid < d) {
        int ai = offset * (2 * thid + 1) - 1;
        int bi = offset * (2 * thid + 2) - 1;
        int t = _input[ai];
        _input[ai] = _input[bi];
        _input[bi] += t;
      }
    }
    __syncwarp();

    if (thid == WARP_SIZE - 1)
      _input[WARP_SIZE] = _input[WARP_SIZE - 1] + last_element;
  }

  typedef struct {
    graph_node_t* set1[UNROLL], * set2[UNROLL], * res[UNROLL];
    graph_node_t set1_size[UNROLL], set2_size[UNROLL], * res_size[UNROLL];
    graph_node_t ub[UNROLL];
    bitarray32 label;
    Graph* g;
  } Arg_t;

  template<bool DIFF>
  __device__ void compute_set(Arg_t* arg) {

    __shared__ int pos[NWARPS_PER_BLOCK][33];

    int wid = threadIdx.x / WARP_SIZE;
    int tid = threadIdx.x % WARP_SIZE;

    for (int i = 0; i < UNROLL; i++) {
      bool still_loop = true;

      int end_pos = 0;

      if (arg->set1_size[i] > 0) {

        for (int idx = tid; (idx < (((arg->set1_size[i] - 1) / WARP_SIZE + 1) * WARP_SIZE) && still_loop); idx += WARP_SIZE) {
          pos[wid][tid] = 0;
          pos[wid][WARP_SIZE] = 0;
          if (idx < arg->set1_size[0] && arg->set1[i][idx] < arg->ub[i]) {
            bitarray32 lb = arg->g->vertex_label[arg->set1[i][idx]];
            if ((lb && arg->label == lb) && (DIFF ^ lower_bound_exist(arg->set2[i], arg->set2_size[i], arg->set1[i][idx]))) {
              pos[wid][tid] = 1;
            }
          }
          else {
            still_loop = false;
          }
          still_loop = __shfl_sync(0xFFFFFFFF, still_loop, 31);

          prefix_sum(&pos[wid][0]);

          graph_node_t res_tmp;
          if (pos[wid][tid + 1] > pos[wid][tid]) {
            res_tmp = arg->set1[i][idx];
          }
          __syncwarp();
          if (pos[wid][tid + 1] > pos[wid][tid]) {
            arg->res[i][end_pos + pos[wid][tid]] = res_tmp;
          }
          end_pos += pos[wid][WARP_SIZE];
        }
      }
      *(arg->res_size[i]) = end_pos;
    }
  }

  __device__ void lock(int* mutex) {
    while (atomicCAS(mutex, 0, 1) != 0);
  }

  __device__ void unlock(int* mutex) {
    atomicExch(mutex, 0);
  }

  __device__ void get_job(JobQueue* q, graph_node_t& cur_pos, graph_node_t& njobs) {
    lock(&(q->mutex));
    cur_pos = q->cur;
    q->cur += JOB_CHUNK_SIZE;
    if (q->cur > q->length) q->cur = q->length;
    njobs = q->cur - cur_pos;
    unlock(&(q->mutex));
  }

  __device__ void extend(Graph* g, Pattern* pat, CallStack* stk, JobQueue* q, pattern_node_t level) {

    __shared__ Arg_t arg[NWARPS_PER_BLOCK];
    int wid = threadIdx.x / WARP_SIZE;

    if (level == 0) {
      graph_node_t cur_job, njobs;

      // TODO: change to warp
      for (int k = 0; k < UNROLL_SIZE(level); k++) {
        if (threadIdx.x % WARP_SIZE == 0) {
          get_job(q, cur_job, njobs);

          for (size_t i = 0; i < njobs; i++) {
            for (int j = 0; j < q->start_level; j++) {
              stk->slot_storage[0][0][k][i + JOB_CHUNK_SIZE * j] = (q->q[cur_job + i].nodes)[j];
            }
          }
          stk->slot_size[0][0][k] = njobs;
          stk->start_level = q->start_level;
        }
        __syncwarp();
      }
    }
    else {

      arg[wid].g = g;

      for (pattern_node_t i = 0; i < PAT_SIZE; i++) {

        if (pat->set_ops[level - 1][i] < 0) break;

        // compute ub based on pattern->partial
        graph_node_t ub = INT_MAX;
        // assert(level >= 1);
        if (i == 0) {
          ub = INT_MAX;
          if (pat->partial[level - 1][i] != 0) {
            for (pattern_node_t k = 0; k < level - 1; k++) {
              if ((pat->partial[level - 1][0] & (1 << k)) && (ub > path(stk, k, stk->uiter[k]))) ub = path(stk, k, stk->uiter[k]);
            }

            for (pattern_node_t k = 0; k < UNROLL_SIZE(level - 1); k++) {
              arg[wid].ub[k] = ub;
              if ((pat->partial[level - 1][0] & (1 << (level - 1))) && (arg[wid].ub[k] > path(stk, level - 1, k))) arg[wid].ub[k] = path(stk, level - 1, k);
            }
          }
          else {
            for (pattern_node_t k = 0; k < UNROLL_SIZE(level - 1); k++) {
              arg[wid].ub[k] = INT_MAX;
            }
          }
        }
        else {
          ub = -1;
          if (pat->partial[level - 1][i] != 0) {
            for (pattern_node_t k = 0; k < level - 1; k++) {
              if ((pat->partial[level - 1][i] & (1 << k)) && (ub < path(stk, k, stk->uiter[k]))) ub = path(stk, k, stk->uiter[k]);
            }
            for (pattern_node_t k = 0; k < UNROLL_SIZE(level - 1); k++) {
              arg[wid].ub[k] = ub;
              if ((pat->partial[level - 1][0] & (1 << (level - 1))) && (arg[wid].ub[k] > path(stk, level - 1, k))) arg[wid].ub[k] = path(stk, level - 1, k);
              if (arg[wid].ub[k] == -1) arg[wid].ub[k] = INT_MAX;
            }
          }
          else {
            for (pattern_node_t k = 0; k < UNROLL_SIZE(level - 1); k++) {
              arg[wid].ub[k] = INT_MAX;
            }
          }
        }

        arg[wid].label = pat->vertex_label[level - 1][i];

        if (pat->set_ops[level - 1][i] & 0x10) {

          graph_node_t* nb = NULL;
          graph_node_t nsize = 0;

          if (level >= 2) {
            nb = &g->colidx[g->rowptr[path(stk, level - 2, stk->uiter[level - 2])]];
            nsize = (graph_node_t)(g->rowptr[path(stk, level - 2, stk->uiter[level - 2]) + 1] - g->rowptr[path(stk, level - 2, stk->uiter[level - 2])]);
          }

          for (graph_node_t k = 0; k < UNROLL_SIZE(level - 1); k++) {
            graph_node_t* neighbor = &g->colidx[g->rowptr[path(stk, level - 1, k)]];
            graph_node_t neighbor_size = (graph_node_t)(g->rowptr[path(stk, level - 1, k) + 1] - g->rowptr[path(stk, level - 1, k)]);
            arg[wid].set1[k] = neighbor;
            arg[wid].set2[k] = nb;
            arg[wid].res[k] = &(stk->slot_storage[level][i][k][0]);
            arg[wid].set1_size[k] = neighbor_size;
            arg[wid].set2_size[k] = nsize;
            arg[wid].res_size[k] = &(stk->slot_size[level][i][k]);
          }
          compute_set<true>(&arg[wid]);


          for (pattern_node_t j = level - 3; j >= 0; j--) {
            nb = &g->colidx[g->rowptr[path(stk, j, stk->uiter[j])]];
            nsize = (graph_node_t)(g->rowptr[path(stk, j, stk->uiter[j]) + 1] - g->rowptr[path(stk, j, stk->uiter[j])]);

            for (graph_node_t k = 0; k < UNROLL_SIZE(level - 1); k++) {
              arg[wid].set1[k] = &(stk->slot_storage[level][i][k][0]);
              arg[wid].set2[k] = nb;
              arg[wid].res[k] = &(stk->slot_storage[level][i][k][0]);
              arg[wid].set1_size[k] = stk->slot_size[level][i][k];
              arg[wid].set2_size[k] = nsize;
              arg[wid].res_size[k] = &(stk->slot_size[level][i][k]);
            }
            compute_set<true>(&arg[wid]);
          }
        }
        else {

          pattern_node_t slot_idx = (pat->set_ops[level - 1][i] & 0xF);

          if (pat->set_ops[level - 1][i] & 0x20) {
            for (graph_node_t k = 0; k < UNROLL_SIZE(level - 1); k++) {
              graph_node_t* neighbor = &g->colidx[g->rowptr[path(stk, level - 1, k)]];
              graph_node_t neighbor_size = (graph_node_t)(g->rowptr[path(stk, level - 1, k) + 1] - g->rowptr[path(stk, level - 1, k)]);

              arg[wid].set1[k] = &(stk->slot_storage[level - 1][slot_idx][k][0]);
              arg[wid].set2[k] = neighbor;
              arg[wid].res[k] = &(stk->slot_storage[level][i][k][0]);
              arg[wid].set1_size[k] = stk->slot_size[level - 1][slot_idx][k];
              arg[wid].set2_size[k] = neighbor_size;
              arg[wid].res_size[k] = &(stk->slot_size[level][i][k]);
            }
            compute_set<false>(&arg[wid]);
          }
          else {

            for (graph_node_t k = 0; k < UNROLL_SIZE(level - 1); k++) {
              graph_node_t* neighbor = &g->colidx[g->rowptr[path(stk, level - 1, k)]];
              graph_node_t neighbor_size = (graph_node_t)(g->rowptr[path(stk, level - 1, k) + 1] - g->rowptr[path(stk, level - 1, k)]);
              arg[wid].set1[k] = &(stk->slot_storage[level - 1][slot_idx][k][0]);
              arg[wid].set2[k] = neighbor;
              arg[wid].res[k] = &(stk->slot_storage[level][i][k][0]);
              arg[wid].set1_size[k] = stk->slot_size[level - 1][slot_idx][k];
              arg[wid].set2_size[k] = neighbor_size;
              arg[wid].res_size[k] = &(stk->slot_size[level][i][k]);
            }
            compute_set<true>(&arg[wid]);
          }
        }
      }
    }
    stk->iter[level] = 0;
    stk->uiter[level] = 0;
  }

  __device__ void match(Graph* g, Pattern* pat, CallStack* stk, JobQueue* q, size_t* count) {
    pattern_node_t level = 0;

    while (true) {

      if (level < pat->nnodes - 1) {

        if (stk->uiter[level] == 0 && stk->slot_size[level][0][0] == 0) {
          extend(g, pat, stk, q, level);
          if (level == 0 && stk->slot_size[level][0][0] == 0) break;
        }

        if (stk->uiter[level] < UNROLL_SIZE(level)) {
          if (stk->iter[level] < stk->slot_size[level][0][stk->uiter[level]]) {
            level++;
          }
          else {
            stk->slot_size[level][0][stk->uiter[level]] = 0;
            stk->iter[level] = 0;
            stk->uiter[level]++;
          }
        }
        else {
          stk->uiter[level] = 0;
          if (level == stk->start_level) {
            level--;
            if (threadIdx.x % WARP_SIZE == 0) {
              // FIXME: fix this for slot size not divisible by UNROLL_SIZE(level)
              stk->iter[0] += UNROLL_SIZE(level);
              for (int j = 1; j < stk->start_level; j++) {
                stk->iter[j] = stk->slot_size[j][0][stk->uiter[level]];
              }
            }
            __syncwarp();
          }
          else if (level > stk->start_level) {
            level--;
            // FIXME: fix this for slot size not divisible by UNROLL_SIZE(level)
            if (threadIdx.x % WARP_SIZE == 0) stk->iter[level] += UNROLL_SIZE(level);
            __syncwarp();
          }
          else if (level > 0) level--;
        }
      }
      else if (level == pat->nnodes - 1) {

        // TODO: we can save the storage of sets for the last level
        extend(g, pat, stk, q, level);
        for (int j = 0; j < UNROLL_SIZE(level); j++) {
          if (threadIdx.x % WARP_SIZE == 0) {
            *count += stk->slot_size[level][0][j];
          }
          __syncwarp();
          stk->slot_size[level][0][j] = 0;
        }
        level--;
        if (threadIdx.x % WARP_SIZE == 0) stk->iter[level] += UNROLL_SIZE(level);
        __syncwarp();
      }
    }
  }

  __global__ void _parallel_match(Graph* dev_graph, Pattern* dev_pattern, CallStack* dev_callstack, JobQueue* job_queue, size_t* res) {

    for (int i = 0;i < dev_graph->nnodes; i++) {
      //printf("%d ", dev_graph->vertex_label[i]);
    }
    __shared__ Graph graph;
    __shared__ Pattern pat;
    __shared__ CallStack stk[NWARPS_PER_BLOCK];
    __shared__ size_t count[NWARPS_PER_BLOCK];

    int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
    int global_wid = global_tid / WARP_SIZE;
    int local_wid = threadIdx.x / WARP_SIZE;

    if (threadIdx.x == 0) {
      graph = *dev_graph;
      pat = *dev_pattern;
    }
    __syncthreads();

    if (threadIdx.x % WARP_SIZE == 0) {
      stk[local_wid] = dev_callstack[global_wid];
    }
    __syncwarp();

    auto start = clock64();
    while (true) {
      match(&graph, &pat, &stk[local_wid], job_queue, &count[local_wid]);
      break;
      // TODO: load balance
    }
    auto stop = clock64();

    if (threadIdx.x % WARP_SIZE == 0) {
      res[global_wid] = count[local_wid];
      // printf("%ld\n", stop - start);
    }
  }
}