#include "hip/hip_runtime.h"
#include "gpu_match.cuh"
#include <hip/hip_runtime.h>

namespace libra {

  // TODO: implement intersection and difference

  template<typename DATA_T, typename SIZE_T>
  __device__
    inline bool lower_bound_exist(DATA_T* set2, SIZE_T set2_size, DATA_T target) {
    int mid;
    int low = 0;
    int high = set2_size;
    while (low < high) {
      mid = low + (high - low) / 2;
      if (target <= set2[mid]) {
        high = mid;
      }
      else {
        low = mid + 1;
      }
    }
    if (low < set2_size && set2[low] < target) {
      low++;
    }
    return (low < set2_size&& set2[low] == target);
  }


  __device__ void prefix_sum(int* _input) {

    int thid = threadIdx.x;
    int offset = 1;
    for (int d = warpSize >> 1; d > 0; d >>= 1) // build sum in place up the tree
    {
      //__syncthreads();
      if (thid < d)
      {
        int ai = offset * (2 * thid + 1) - 1;
        int bi = offset * (2 * thid + 2) - 1;
        _input[bi] += _input[ai];
      }
      offset *= 2;
    }
    if (thid == 0) { _input[warpSize - 1] = 0; } // clear the last element
    for (int d = 1; d < warpSize; d *= 2) // traverse down tree & build scan
    {
      offset >>= 1;
      //__syncthreads();
      if (thid < d)
      {
        int ai = offset * (2 * thid + 1) - 1;
        int bi = offset * (2 * thid + 2) - 1;
        int t = _input[ai];
        _input[ai] = _input[bi];
        _input[bi] += t;
      }
    }
    //__syncthreads();
  }


  template<typename DATA_T, typename SIZE_T>
  __device__ void intersection(DATA_T* set1, DATA_T* set2, DATA_T* _res, SIZE_T set1_size, SIZE_T set2_size, SIZE_T* res_size, DATA_T ub) {

    __shared__ int pos[BLOCK_DIM];

    int tid = threadIdx.x;
    int end_pos = 0;
    int loop_end = set1_size - set1_size % warpSize + warpSize;
    bool last_find;
    for (int idx = tid; idx < loop_end && set1[idx] < ub; idx += warpSize) {
      last_find = false;
      pos[tid] = 0;
      if (idx < set1_size && set1[idx] < ub) {
        if (lower_bound_exist(set2, set2_size, set1[idx])) {
          pos[tid] = 1;
        }
        if (threadIdx.x == warpSize - 1 && pos[warpSize - 1] == 1) {
          last_find = true;
        }
      }
      prefix_sum(pos);

      if ((tid == warpSize - 1 && last_find) ||
        tid != warpSize - 1 && pos[tid] < pos[tid + 1]) {
        _res[end_pos + pos[tid]] = set1[idx];
      }
      end_pos += pos[warpSize - 1];
    }
    // __syncthreads();
     /*
     if(tid==0){
       for(int i =0; i<end_pos; i++){
         printf("%d ", _res[i]);
       }
       printf("\n");
     }
 */
    *res_size = end_pos;
  }

  template<typename DATA_T, typename SIZE_T>
  __device__ void difference(DATA_T* set1, DATA_T* set2, DATA_T* res, SIZE_T set1_size, SIZE_T set2_size, SIZE_T* res_size, DATA_T ub) {

    __shared__ int pos[BLOCK_DIM];

    int tid = threadIdx.x;
    int end_pos = 0;
    int loop_end = set1_size - set1_size % warpSize + warpSize;
    bool last_find;
    for (int idx = tid; idx < loop_end; idx += warpSize) {
      last_find = false;
      pos[tid] = 0;

      if (idx < set1_size && set1[idx] < ub) {
        if (!lower_bound_exist(set2, set2_size, set1[idx])) {
          pos[tid] = 1;
        }
        if (threadIdx.x == warpSize - 1 && pos[warpSize - 1] == 1) {
          last_find = true;
        }
      }
      prefix_sum(pos);

      if ((tid == warpSize - 1 && last_find) ||
        tid != warpSize - 1 && pos[tid] < pos[tid + 1]) {
        res[end_pos + pos[tid]] = set1[idx];
      }
      end_pos += pos[warpSize - 1];
    }
    // __syncthreads();
     /*
     if(tid==0){
       for(int i =0; i<end_pos; i++){
         printf("%d ", _res[i]);
       }
       printf("\n");
     }
 */
 //clock_t stop_time = clock();
 //if(threadIdx.x==0){
     //printf("time pointer2:%p\n", time_);
     //time_[blockIdx.x] = stop_time - start_time;
 //}
    * res_size = end_pos;

  }

  __device__ void lock(int* mutex) {
    while (atomicCAS(mutex, 0, 1) != 0);
  }

  __device__ void unlock(int* mutex) {
    atomicExch(mutex, 0);
  }

  __device__ void get_job(JobQueue* q, graph_node_t& cur_job, graph_node_t& njobs) {
    lock(&(q->mutex));
    graph_node_t cur_pos = q->cur;
    q->cur += JOB_CHUNK_SIZE;
    if (q->cur > q->length) q->cur = q->length;
    njobs = q->cur - cur_pos;
    unlock(&(q->mutex));
  }

  __device__ void match(Graph* g, Pattern* pat, CallStack* stk, JobQueue* q, size_t* count) {
    graph_node_t level = 0;

    while (true) {

      if (level < pat->nnodes - 1) {
        if (stk->slot_size[level][0] == 0) {

          // extend 
          if (level == 0) {
            graph_node_t cur_job, njobs;

            // TODO: change to warp
            if (threadIdx.x % WARP_SIZE == 0) {
              get_job(q, cur_job, njobs);
              stk->iter[0] = 0;
              for (size_t i = 0; i < njobs; i++) {
                stk->slot_storage[0][0][i] = q->queue[cur_job + i];
              }
              stk->slot_size[0][0] = njobs;
            }
            __syncwarp();
          }
          else {

            for (pattern_node_t i = 0; i < PAT_SIZE; i++) {

              // TODO: compute ub based on pattern->partial
              graph_node_t ub = INT_MAX;
              if (pat->partial[level - 1][i] >= 0) ub = stk->path[pat->partial[level - 1][i]];

              graph_node_t* neighbor = g->rowptr[stk->path[level - 1]] + g->colidx;
              graph_node_t neighbor_size = (graph_node_t)(g->rowptr[stk->path[level - 1] + 1] - g->rowptr[stk->path[level - 1]]);

              if (pat->set_ops[level - 1][i] & 0x10) {

                graph_node_t* nb = NULL;
                graph_node_t nsize = 0;

                if (level >= 2) {
                  nb = g->rowptr[stk->path[level - 2]] + g->colidx;
                  nsize = (graph_node_t)(g->rowptr[stk->path[level - 2] + 1] - g->rowptr[stk->path[level - 2]]);
                }
                // when the second set is empty, the difference function simply checks ub and copy first set to res set.
                difference(neighbor, nb, &(stk->slot_storage[level][i][0]), neighbor_size, nsize, &(stk->slot_size[level][i]), ub);

                for (pattern_node_t j = level - 3; j >= 0; j--) {
                  nb = g->rowptr[stk->path[j]] + g->colidx;
                  nsize = (graph_node_t)(g->rowptr[stk->path[j] + 1] - g->rowptr[stk->path[j]]);

                  difference(&(stk->slot_storage[level][i][0]), nb, &(stk->slot_storage[level][i][0]), stk->slot_size[level][i], nsize, &(stk->slot_size[level][i]), ub);
                }
              }
              else {

                pattern_node_t slot_idx = (pat->set_ops[level - 1][i] & 0xF);

                if (pat->set_ops[level - 1][i] & 0x20) {
                  intersection(&(stk->slot_storage[level - 1][slot_idx][0]), neighbor, &(stk->slot_storage[level][i][i]), stk->slot_size[level - 1][slot_idx], neighbor_size, &(stk->slot_size[level][i]), ub);
                }
                else {
                  difference(&(stk->slot_storage[level - 1][slot_idx][0]), neighbor, &(stk->slot_storage[level][i][i]), stk->slot_size[level - 1][slot_idx], neighbor_size, &(stk->slot_size[level][i]), ub);
                }
              }

              if (pat->set_ops[level - 1][i] < 0) break;
            }
          } //end extend

          stk->iter[level] = 0;
        }
        if (stk->iter[level] < stk->slot_size[level][0]) {
          stk->path[level] = stk->slot_storage[level][0][stk->iter[level]];
          level++;
        }
        else {
          stk->slot_size[level][0] = 0;
          level--;
          if (level < 0) {
            break;
          }
          if (threadIdx.x % WARP_SIZE == 0) stk->iter[level]++;
          __syncwarp();
        }
      }
      else if (level == pat->nnodes - 1) {
        // TODO: we can save the storage of sets for the last level
        if (threadIdx.x % WARP_SIZE == 0) {
          count += stk->slot_size[level][0];
        }
        __syncwarp();
        stk->slot_size[level][0] = 0;
        level--;
      }
    }
  }

  __global__ void _parallel_match(Graph* dev_graph, Pattern* dev_pattern, CallStack* dev_callstack, JobQueue* job_queue) {

    __shared__ Graph graph;
    __shared__ Pattern pat;
    __shared__ JobQueue queue;
    __shared__ CallStack stk[NWARPS_PER_BLOCK];
    __shared__ size_t count[NWARPS_PER_BLOCK];

    int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
    int global_wid = global_tid / WARP_SIZE;
    int local_wid = threadIdx.x / WARP_SIZE;

    // TODO: change to warp 
    // graph and pattern are shared among all threads in a threadblock
    if (threadIdx.x == 0) {
      graph = *dev_graph;
      pat = *dev_pattern;
      queue = *job_queue;
    }
    __syncwarp();

    // TODO: change to warp
    // callstack is shared among all threads in each warp
    if (threadIdx.x % WARP_SIZE == 0) {
      stk[local_wid] = dev_callstack[global_wid];
    }
    __syncwarp();

    while (true) {
      match(&graph, &pat, &stk[local_wid], &queue, &count[local_wid]);
      break;
      // TODO: load balance
    }
  }
}