#include "hip/hip_runtime.h"
#include "gpu_match.cuh"
#include "trans_stk.h"
#include <hip/hip_runtime.h>

// TODO: change this to gpu array so we can use different unroll for diffrent levels
#define UNROLL_SIZE(l) UNROLL

namespace libra {

  inline __device__ graph_node_t path(CallStack* stk, Pattern* pat, int level, int k) {
    if (level > 0)
      return stk->slot_storage[pat->rowptr[level]][stk->uiter[level]][stk->iter[level] + k];
    else {
      return stk->slot_storage[0][stk->uiter[0]][stk->iter[0] + k + (stk->start_level + level - 1) * JOB_CHUNK_SIZE];
    }
  }

  typedef struct {
    graph_node_t* set1[UNROLL], * set2[UNROLL], * res[UNROLL];
    graph_node_t set1_size[UNROLL], set2_size[UNROLL], * res_size[UNROLL];
    graph_node_t ub[UNROLL];
    bitarray32 label;
    Graph* g;
    int num_sets;
  } Arg_t;

  template<typename DATA_T, typename SIZE_T>
  inline __device__
    bool bsearch_exist(DATA_T* set2, SIZE_T set2_size, DATA_T target) {
    int mid;
    int low = 0;
    int high = set2_size - 1;
    while (low <= high) {
      mid = (low + high) / 2;
      if (target == set2[mid]) {
        return true;
      }
      else if (target > set2[mid]) {
        low = mid + 1;
      }
      else {
        high = mid - 1;
      }
    }
    return false;
  }

  template<typename DATA_T, typename SIZE_T>
  inline __device__
    SIZE_T upper_bound(DATA_T* set2, SIZE_T set2_size, DATA_T target) {
    int i, step;
    int low = 0;
    while (set2_size > 0) {
      i = low;
      step = set2_size / 2;
      i += step;
      if (target > set2[i]) {
        low = ++i; set2_size -= step + 1;
      }
      else {
        set2_size = step;
      }
    }
    return low;
  }

  inline __device__
    void prefix_sum(int* _input, int input_size) {

    int thid = threadIdx.x % WARP_SIZE;
    int offset = 1;
    int last_element = _input[input_size - 1];
    // build sum in place up the tree
    for (int d = (WARP_SIZE >> 1); d > 0; d >>= 1) {
      if (thid < d) {
        int ai = offset * (2 * thid + 1) - 1;
        int bi = offset * (2 * thid + 2) - 1;
        _input[bi] += _input[ai];
      }
      offset <<= 1;
    }
    if (thid == 0) { _input[WARP_SIZE - 1] = 0; } // clear the last element
     // traverse down tree & build scan
    for (int d = 1; d < WARP_SIZE; d <<= 1) {
      offset >>= 1;
      if (thid < d) {
        int ai = offset * (2 * thid + 1) - 1;
        int bi = offset * (2 * thid + 2) - 1;
        int t = _input[ai];
        _input[ai] = _input[bi];
        _input[bi] += t;
      }
    }
    __syncwarp();

    if (thid >= input_size - 1)
      _input[thid + 1] = _input[input_size - 1] + last_element;
  }


  template<bool DIFF>
  __device__ void compute_set(Arg_t* arg) {

    __shared__ int pos[NWARPS_PER_BLOCK][WARP_SIZE + 1];
    __shared__ graph_node_t size_psum[NWARPS_PER_BLOCK][WARP_SIZE + 1];
    __shared__ int end_pos[NWARPS_PER_BLOCK][UNROLL];

    int wid = threadIdx.x / WARP_SIZE;
    int tid = threadIdx.x % WARP_SIZE;


    if (tid < arg->num_sets) {
      arg->set1_size[tid] = upper_bound(arg->set1[tid], arg->set1_size[tid], arg->ub[tid]);
      size_psum[wid][tid] = arg->set1_size[tid];
      end_pos[wid][tid] = 0;
    }
    else {
      size_psum[wid][tid] = 0;
    }
    __syncwarp();

    prefix_sum(&size_psum[wid][0], arg->num_sets);
    __syncwarp();


    bool still_loop = true;
    int slot_idx = 0;
    int offset = 0;

    int size1 = (size_psum[wid][WARP_SIZE] > 0) ? (((size_psum[wid][WARP_SIZE] - 1) / WARP_SIZE + 1) * WARP_SIZE) : 0;

    for (int idx = tid; (idx < size1 && still_loop); idx += WARP_SIZE) {
      pos[wid][tid] = 0;

      if (idx < size_psum[wid][WARP_SIZE]) {

        while (idx >= size_psum[wid][slot_idx + 1]) {
          slot_idx++;
        }
        offset = idx - size_psum[wid][slot_idx];

        bitarray32 lb = arg->g->vertex_label[arg->set1[slot_idx][offset]];
        if ((lb && arg->label == lb) && (DIFF ^ bsearch_exist(arg->set2[slot_idx], arg->set2_size[slot_idx], arg->set1[slot_idx][offset]))) {
          pos[wid][tid] = 1;
        }
      }
      else {
        slot_idx = arg->num_sets;
        still_loop = false;
      }

      still_loop = __shfl_sync(0xFFFFFFFF, still_loop, 31);

      prefix_sum(&pos[wid][0], WARP_SIZE);
      __syncwarp();

      graph_node_t res_tmp;
      if (pos[wid][tid + 1] > pos[wid][tid]) {
        res_tmp = arg->set1[slot_idx][offset];
      }
      __syncwarp();
      int prev_idx = ((idx / WARP_SIZE == size_psum[wid][slot_idx] / WARP_SIZE) ? size_psum[wid][slot_idx] % WARP_SIZE : 0);

      if (pos[wid][tid + 1] > pos[wid][tid]) {
        arg->res[slot_idx][end_pos[wid][slot_idx] + pos[wid][tid] - pos[wid][prev_idx]] = res_tmp;
      }

      if (slot_idx < __shfl_down_sync(0xFFFFFFFF, slot_idx, 1)) {
        end_pos[wid][slot_idx] += pos[wid][tid + 1] - pos[wid][prev_idx];
      }
      else if (tid == WARP_SIZE - 1 && slot_idx < arg->num_sets) {
        end_pos[wid][slot_idx] += pos[wid][WARP_SIZE] - pos[wid][prev_idx];
      }
    }
    __syncwarp();
    if (tid < arg->num_sets) {
      *(arg->res_size[tid]) = end_pos[wid][tid];
    }
    __syncwarp();
  }

  inline __device__ void get_job(JobQueue* q, graph_node_t& cur_pos, graph_node_t& njobs) {
    lock(&(q->mutex));
    cur_pos = q->cur;
    q->cur += JOB_CHUNK_SIZE;
    if (q->cur > q->length) q->cur = q->length;
    njobs = q->cur - cur_pos;
    unlock(&(q->mutex));
    // cur_pos = atomicAdd(&q->cur, JOB_CHUNK_SIZE);
    // if (cur_pos < q->length) {
    //   njobs = (cur_pos + JOB_CHUNK_SIZE > q->length) ?  q->length - cur_pos : JOB_CHUNK_SIZE;
    // }
    // else njobs = 0;
  }

  __device__ void extend(Graph* g, Pattern* pat, CallStack* stk, JobQueue* q, pattern_node_t level) {

    __shared__ Arg_t arg[NWARPS_PER_BLOCK];
    int wid = threadIdx.x / WARP_SIZE;

    if (level == 0) {
      graph_node_t cur_job, njobs;

      // TODO: change to warp
      for (int k = 0; k < UNROLL_SIZE(level); k++) {
        if (threadIdx.x % WARP_SIZE == 0) {
          get_job(q, cur_job, njobs);

          for (size_t i = 0; i < njobs; i++) {
            for (int j = 0; j < q->start_level; j++) {
              stk->slot_storage[0][k][i + JOB_CHUNK_SIZE * j] = (q->q[cur_job + i].nodes)[j];
            }
          }
          stk->slot_size[0][k] = njobs;
          stk->start_level = q->start_level;
        }
        __syncwarp();
      }
    }
    else {

      arg[wid].g = g;
      arg[wid].num_sets = UNROLL_SIZE(level);

      int remaining = stk->slot_size[pat->rowptr[level - 1]][stk->uiter[level - 1]] - stk->iter[level - 1];
      if (remaining >= 0 && UNROLL_SIZE(level - 1) > remaining) {
        arg[wid].num_sets = remaining;
      }

      for (int i = pat->rowptr[level]; i < pat->rowptr[level + 1]; i++) {

        // compute ub based on pattern->partial
        graph_node_t ub = ((i == pat->rowptr[level]) ? INT_MAX : -1);
        if (pat->partial[i] != 0) {

          // compute ub with nodes after start_level until previous level
          for (pattern_node_t k = 1; k < level - 1; k++) {
            if ((pat->partial[i] & (1 << (k + stk->start_level - 1))) && ((i == pat->rowptr[level]) ^ (ub < path(stk, pat, k, stk->uiter[k + 1])))) ub = path(stk, pat, k, stk->uiter[k + 1]);
          }
          // compute ub with nodes in the previous level
          for (pattern_node_t k = 0; k < arg[wid].num_sets; k++) {
            arg[wid].ub[k] = ub;
            int prev_level = (level > 1 ? stk->start_level : stk->start_level - 1);
            int prev_iter = (level > 1 ? stk->uiter[1] : k);
            // compute ub with the first few nodes before start_level
            for (pattern_node_t j = 0; j < prev_level; j++) {
              if ((pat->partial[i] & (1 << j)) && ((i == pat->rowptr[level]) ^ (arg[wid].ub[k] < path(stk, pat, j + 1 - stk->start_level, prev_iter)))) arg[wid].ub[k] = path(stk, pat, j + 1 - stk->start_level, prev_iter);
            }

            if ((pat->partial[i] & (1 << (level + stk->start_level - 2))) && ((i == pat->rowptr[level]) ^ (arg[wid].ub[k] < path(stk, pat, level - 1, k)))) arg[wid].ub[k] = path(stk, pat, level - 1, k);
            if (arg[wid].ub[k] == -1) arg[wid].ub[k] = INT_MAX;
          }
        }
        else {
          for (pattern_node_t k = 0; k < arg[wid].num_sets; k++) {
            arg[wid].ub[k] = INT_MAX;
          }
        }

        arg[wid].label = pat->slot_labels[i];

        if (pat->set_ops[i] & 0x20) {

          for (graph_node_t k = 0; k < arg[wid].num_sets; k++) {

            arg[wid].set2[k] = NULL;
            arg[wid].set2_size[k] = 0;

            if (!EDGE_INDUCED) {
              graph_node_t t = path(stk, pat, level - 2, ((level > 1) ? stk->uiter[level - 1] : k));
              arg[wid].set2[k] = &g->colidx[g->rowptr[t]];
              arg[wid].set2_size[k] = (graph_node_t)(g->rowptr[t + 1] - g->rowptr[t]);
            }
            graph_node_t t = path(stk, pat, level - 1, k);
            arg[wid].set1[k] = &g->colidx[g->rowptr[t]];
            arg[wid].res[k] = &(stk->slot_storage[i][k][0]);
            arg[wid].set1_size[k] = (graph_node_t)(g->rowptr[t + 1] - g->rowptr[t]);
            arg[wid].res_size[k] = &(stk->slot_size[i][k]);
          }
          compute_set<true>(&arg[wid]);

          if (!EDGE_INDUCED) {
            for (pattern_node_t j = level - 3; j >= 1 - stk->start_level; j--) {
              graph_node_t t = path(stk, pat, j, stk->uiter[(j > 0 ? j + 1 : 1)]);
              
              for (graph_node_t k = 0; k < arg[wid].num_sets; k++) {
                arg[wid].set1[k] = &(stk->slot_storage[i][k][0]);
                arg[wid].set2[k] = &g->colidx[g->rowptr[t]];
                arg[wid].res[k] = &(stk->slot_storage[i][k][0]);
                arg[wid].set1_size[k] = stk->slot_size[i][k];
                arg[wid].set2_size[k] = (graph_node_t)(g->rowptr[t + 1] - g->rowptr[t]);
                arg[wid].res_size[k] = &(stk->slot_size[i][k]);
              }
              compute_set<true>(&arg[wid]);
            }
          }
          for (graph_node_t k = arg[wid].num_sets; k < UNROLL_SIZE(level); k++) stk->slot_size[i][k] = 0;
        }
        else {

          pattern_node_t slot_idx = (pat->set_ops[i] & 0x1F);

          if (pat->set_ops[i] & 0x40) {
            for (graph_node_t k = 0; k < arg[wid].num_sets; k++) {
              graph_node_t t = path(stk, pat, level - 1, k);
              graph_node_t* neighbor = &g->colidx[g->rowptr[t]];
              graph_node_t neighbor_size = (graph_node_t)(g->rowptr[t + 1] - g->rowptr[t]);

              if (level > 1) {
                arg[wid].set1[k] = &(stk->slot_storage[slot_idx][stk->uiter[level - 1]][0]);
                arg[wid].set1_size[k] = stk->slot_size[slot_idx][stk->uiter[level - 1]];
              }
              else {
                graph_node_t t = path(stk, pat, -1, k);
                arg[wid].set1[k] = &g->colidx[g->rowptr[t]];
                arg[wid].set1_size[k] = (graph_node_t)(g->rowptr[t + 1] - g->rowptr[t]);
              }

              arg[wid].set2[k] = neighbor;
              arg[wid].set2_size[k] = neighbor_size;
              arg[wid].res[k] = &(stk->slot_storage[i][k][0]);
              arg[wid].res_size[k] = &(stk->slot_size[i][k]);
            }
            compute_set<false>(&arg[wid]);
            for (graph_node_t k = arg[wid].num_sets; k < UNROLL_SIZE(level); k++) stk->slot_size[i][k] = 0;

          }
          else {

            for (graph_node_t k = 0; k < arg[wid].num_sets; k++) {
              graph_node_t* neighbor = NULL;
              graph_node_t neighbor_size = 0;
              if (!EDGE_INDUCED) {
                graph_node_t t = path(stk, pat, level - 1, k);
                neighbor = &g->colidx[g->rowptr[t]];
                neighbor_size = (graph_node_t)(g->rowptr[t + 1] - g->rowptr[t]);
              }

              if (level > 1) {
                arg[wid].set1[k] = &(stk->slot_storage[slot_idx][stk->uiter[level - 1]][0]);
                arg[wid].set1_size[k] = stk->slot_size[slot_idx][stk->uiter[level - 1]];
              }
              else {
                graph_node_t t = path(stk, pat, -1, k);
                arg[wid].set1[k] = &g->colidx[g->rowptr[t]];
                arg[wid].set1_size[k] = (graph_node_t)(g->rowptr[t + 1] - g->rowptr[t]);
              }

              arg[wid].set2[k] = neighbor;
              arg[wid].set2_size[k] = neighbor_size;
              arg[wid].res[k] = &(stk->slot_storage[i][k][0]);
              arg[wid].res_size[k] = &(stk->slot_size[i][k]);
            }
            compute_set<true>(&arg[wid]);
            for (graph_node_t k = arg[wid].num_sets; k < UNROLL_SIZE(level); k++) stk->slot_size[i][k] = 0;

          }
        }
      }
    }
    stk->iter[level] = 0;
    stk->uiter[level] = 0;
  }

  __device__ void match(Graph* g, Pattern* pat, CallStack* stk, JobQueue* q, size_t* count, int* mutex) {
    //pattern_node_t level = 0;
    //printf("stk->start_level:%d\n", stk->start_level);
    pattern_node_t& level = stk->level;
    while (true) {
      if (threadIdx.x % WARP_SIZE == 0){
        lock(mutex);
      }
      __syncwarp();

      if (level < pat->nnodes - stk->start_level) {

        if (stk->uiter[level] == 0 && stk->slot_size[pat->rowptr[level]][0] == 0) {
          extend(g, pat, stk, q, level);
          if (level == 0 && stk->slot_size[0][0] == 0) {
              if (threadIdx.x % WARP_SIZE == 0) unlock(mutex);
              __syncwarp();
              break;
          }
        }

        if (stk->uiter[level] < UNROLL_SIZE(level)) {
          if (stk->iter[level] < stk->slot_size[pat->rowptr[level]][stk->uiter[level]]) {
            if (threadIdx.x % WARP_SIZE == 0) level++;
            __syncwarp();
          }
          else {
            stk->slot_size[pat->rowptr[level]][stk->uiter[level]] = 0;
            stk->iter[level] = 0;
             if (threadIdx.x % WARP_SIZE == 0) stk->uiter[level]++;
             __syncwarp();
          }
        }
        else {
          stk->uiter[level] = 0;
          if (level > 0) {
            if (threadIdx.x % WARP_SIZE == 0) level--;
            if (threadIdx.x % WARP_SIZE == 0) stk->iter[level] += UNROLL_SIZE(level + 1);
            __syncwarp();
          }
        }
      }
      else if (level == pat->nnodes - stk->start_level) {

        // TODO: we can save the storage of sets for the last level
        extend(g, pat, stk, q, level);
        for (int j = 0; j < UNROLL_SIZE(level); j++) {
          if (threadIdx.x % WARP_SIZE == 0) {
            *count += stk->slot_size[pat->rowptr[level]][j];
          }
          __syncwarp();
          stk->slot_size[pat->rowptr[level]][j] = 0;
        }
        if (threadIdx.x % WARP_SIZE == 0) level--;
        if (threadIdx.x % WARP_SIZE == 0) stk->iter[level] += UNROLL_SIZE(level + 1);
        __syncwarp();
      }
      //__syncwarp();
      if (threadIdx.x % WARP_SIZE == 0) unlock(mutex);
      __syncwarp();
    }
  }

  __global__ void _parallel_match(Graph* dev_graph, Pattern* dev_pattern, CallStack* dev_callstack, JobQueue* job_queue, size_t* res) {

    __shared__ Graph graph;
    __shared__ Pattern pat;
    __shared__ CallStack stk[NWARPS_PER_BLOCK];
    __shared__ size_t count[NWARPS_PER_BLOCK];

    __shared__ bool stealed[NWARPS_PER_BLOCK];

    __shared__ int mutex_this_block[NWARPS_PER_BLOCK];

    int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
    int global_wid = global_tid / WARP_SIZE;
    int local_wid = threadIdx.x / WARP_SIZE;

    if (threadIdx.x == 0) {
      graph = *dev_graph;
      pat = *dev_pattern;
    }
    __syncthreads();

    if (threadIdx.x % WARP_SIZE == 0) {
      stk[local_wid] = dev_callstack[global_wid];
    }
    __syncwarp();

    auto start = clock64();
    
    while (true) {
      __syncwarp();
      match(&graph, &pat, &stk[local_wid], job_queue, &count[local_wid], &mutex_this_block[local_wid]);
      __syncwarp();
      //break;

      //trans_success[local_wid]=false;
      if(threadIdx.x % WARP_SIZE == 0){
        stealed[local_wid] = trans_skt(stk,  &stk[local_wid],  &pat, mutex_this_block);
      }

       __syncwarp();

      //break;
      
     
      if(stealed[local_wid]){
        //printf("stealed_idx:%d\n", stealed_idx[local_wid]);
        //break;
         continue;
      }
      else{
          break;
      }
    }

    auto stop = clock64();
   

    if (threadIdx.x % WARP_SIZE == 0) {
      res[global_wid] = count[local_wid];
      //printf("%d\t%ld\t%d\t%d\n", blockIdx.x, stop - start, stealed[local_wid], local_wid);
      //printf("%ld\n", stop - start);
    }
    
    //if(threadIdx.x % WARP_SIZE == 0)
    //  printf("%d\t%d\t%d\n", blockIdx.x, local_wid, mutex_this_block[local_wid]);
    
  }
}
